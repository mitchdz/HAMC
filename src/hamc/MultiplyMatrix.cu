#include "hip/hip_runtime.h"

#ifndef HAMC_MULTIPLY_MATRIX_H
#define HAMC_MULTIPLY_MATRIX_H

#include <stdio.h>
#include <hip/hip_runtime.h>
//#include <cuda/pipeline>
#include "hamc_common.h"

//#define TILE_WIDTH 16

//int TILE_WIDTH = 16;

/*__global__ void mult_kernel_outer_product(HAMC_DATA_TYPE_t *A, HAMC_DATA_TYPE_t *B, HAMC_DATA_TYPE_t *C, int rowA, int rowB, int colA, int colB, int TILE_WIDTH)
{
    extern __shared__ HAMC_DATA_TYPE_t sharedA[];
    
    //int TILE_WIDTH = (sizeof(sharedArray) / sizeof(sharedArray[0])) / 4;
    
    //HAMC_DATA_TYPE_t *sharedA = sharedArray;
    //HAMC_DATA_TYPE_t *sharedB = &sharedA[TILE_WIDTH * TILE_WIDTH];
    //extern __shared__ HAMC_DATA_TYPE_t sharedA[];
    //extern __shared__ HAMC_DATA_TYPE_t sharedB[];
    
    int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    int tid = threadIdx.y * TILE_WIDTH + threadIdx.x;
    int tilePos = 0;

    HAMC_DATA_TYPE_t b = 0;
    HAMC_DATA_TYPE_t pValue = 0;
  
    for(int i = 0; (i < ((colA - 1)/TILE_WIDTH) + 1) && (i < ((rowB - 1)/TILE_WIDTH) + 1); i++){
        tilePos = i * TILE_WIDTH;
        if((Row < rowA) && (tilePos + threadIdx.x < colA)){
            sharedA[tid] = A[Row * colA + tilePos + threadIdx.x];
        }
        else{
            sharedA[tid] = 0;
        }
        if((Col < colB) && (tilePos + threadIdx.y < rowB)){
            b = B[(tilePos + threadIdx.y) * colB + Col];
        }
        else{
            b = 0;
        }
        __syncthreads();
        
        if((Row < rowA) && (Col < colB)){
            for(int j = 0; j < TILE_WIDTH; j++){
                b = B[(j * colB) +  * (blockDim.x * blockDim.y)];
                pValue ^= (sharedA[threadIdx.y * TILE_WIDTH + j] & b);
            }
        }
        
        __syncthreads();
    }
    if((Row < rowA) && (Col < colB)){
        C[Row * colB + Col] = pValue;
    }
}/**/

/*__global__ void mult_kernel_register_blocked(HAMC_DATA_TYPE_t *A, HAMC_DATA_TYPE_t *B, HAMC_DATA_TYPE_t *C, int rowA, int rowB, int colA, int colB)
{
    __shared__ HAMC_DATA_TYPE_t sharedA[64*64];
    __shared__ HAMC_DATA_TYPE_t sharedB[64*64];
    
    int tile = 64;
    HAMC_DATA_TYPE_t regC[16];
    
    int tileRow = blockIdx.y * tile;
    int tileCol = blockIdx.x * tile;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;//0-255
    int tilePosX = tid % tile;//0-63
    int tilePosY = tid / tile;//0-3
    int row = 0, sharedIndex = 0;
    int stride = (blockDim.x * blockDim.y) / tile;
    for(int i = 0; (i < ((colA - 1)/tile) + 1) && (i < ((rowB - 1)/tile) + 1) ; i++){
        for(int j = 0; j < (tile * tile) / (blockDim.x * blockDim.y); j++){//0-15
            row = j * stride + tilePosY;
            sharedIndex = row * tile + tilePosX;
            if(((tileRow + row) < rowA) && (((i * tile) + tilePosX) < colA)){
                sharedA[sharedIndex] = A[((tileRow + row) * colA) + (i * tile) + tilePosX];
            }
            else{
                sharedA[sharedIndex] = 0;
            }
            if(((row + (i * tile)) < rowB) && ((tileCol + tilePosX) < colB)){
                sharedB[sharedIndex] = B[(colB * (row + (i * tile))) + tileCol + tilePosX];
            }
            else{
                sharedB[sharedIndex] = 0;
            }
        }
        __syncthreads();
        for(int j = 0; j < blockDim.x; j++){
            //stride = j * blockDim.x / tile;
            for(int k = 0; k < tile; k++){
                regC[j] ^= sharedA[((j * blockDim.x / tile) + (tid % blockDim.x)) * tile + k] & sharedB[(k * tile) + (j * blockDim.x / tile) + (tid % blockDim.x)];
            }
        }
        __syncthreads();
    }
    __syncthreads();
    for(int i = 0; i < blockDim.x; i++){
        C[(tileRow + tid % blockDim.x + blockDim.y * (i / stride) * colB) + tileCol + tid % tile] = regC[i];
    }
}/**/

__global__ void mult_kernel_compressed_data(HAMC_DATA_TYPE_t *A, HAMC_DATA_TYPE_t *B, HAMC_DATA_TYPE_t *C, int rowA, int rowB, int colA, int colB, int TILE_WIDTH)
{
    extern __shared__ HAMC_DATA_TYPE_t sharedArray[];
    
    HAMC_DATA_TYPE_t *sharedA = sharedArray;
    uint32_t *sharedFloatA = (uint32_t *)sharedA;
    uint32_t *sharedFloatB = &sharedFloatA[TILE_WIDTH * TILE_WIDTH];
    HAMC_DATA_TYPE_t *sharedB = (uint8_t *)sharedFloatB;
    //uint32_t *transposeFloatB = &sharedFloatB[TILE_WIDTH * TILE_WIDTH];
    //HAMC_DATA_TYPE_t *transposeB = (uint8_t *)transposeFloatB;
    
    uint32_t *floatA = (uint32_t *)A;
    uint32_t *floatB = (uint32_t *)B;
    
    int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    int tid = threadIdx.y * TILE_WIDTH + threadIdx.x;
    int tilePos = 0;
    
    //HAMC_DATA_TYPE_t pValue[4];
    //uint32_t *pValueFloat = (uint32_t *)pValue;
    uint32_t pValueFloat = 0;
    HAMC_DATA_TYPE_t shortValue = 0;
    
    //if(blockIdx.x == 0 && blockIdx.y == 0 && tid == 0) printf("1");
    //if(blockIdx.x == 0 && blockIdx.y == 0){
    for(int i = 0; i < ((colA - 1)/(TILE_WIDTH * 4)) + 1; i++){
        tilePos = i / 4 * TILE_WIDTH;
        if((Row < rowA) && (tilePos + threadIdx.x < colA)){
            sharedFloatA[tid] = floatA[Row * colA / 4 + tilePos + threadIdx.x];
        }
        else{
            sharedFloatA[tid] = (uint32_t)0;
        }
        //sharedFloatB[tid] = floatB[(threadIdx.y * 4 + threadIdx.x / 4) * colB + tilePos + threadIdx.x % 8];
        __syncthreads();
        
        /*for(int j = 0; j < 4; j++){
            //transposeB[] = sharedB[];
            transposeB[((threadIdx.x * 4 + j * TILE_WIDTH) * TILE_WIDTH) + threadIdx.y] = sharedB[threadIdx.y * 4 * TILE_WIDTH + threadIdx.x + j * TILE_WIDTH];
            //transposeB[(threadIdx.x * 4 + j) * TILE_WIDTH + threadIdx.y * 4] = sharedB[(threadIdx.y * 4 * TILE_WIDTH) + threadIdx.x * 4 + j];
            //transposeB[threadIdx.x * TILE_WIDTH + threadIdx.y * 4 + j] = sharedB[(threadIdx.y * 4 + j) * TILE_WIDTH + threadIdx.x];
        }/**/
        for(int j = 0; j < 4; j++){
            //sharedB[] = B[];
            if(((threadIdx.y + (j + tilePos * 4) * TILE_WIDTH) < rowB) && (Col < colB)){
                sharedB[threadIdx.x * 4 * TILE_WIDTH + j * TILE_WIDTH + threadIdx.y] = B[colB * (threadIdx.y + (j + tilePos * 4) * TILE_WIDTH) + Col];
            }
            else{
                sharedB[threadIdx.x * 4 * TILE_WIDTH + j * TILE_WIDTH + threadIdx.y] = (uint8_t)0;
            }
            //sharedB[tid * 4 + j] = B[(j + ((tilePos + threadIdx.y) * 4)) * colB + Col];
            //sharedB[threadIdx.x * 4 * TILE_WIDTH + threadIdx.y * 4 + j] = B[(j + ((tilePos + threadIdx.y) * 4)) * colB + Col];
        }/**/
        __syncthreads();
        
        if(blockIdx.x == 0 && blockIdx.y == 0 && tid == 0){// && i == 0){
            printf("A: i = %d\n", i);
            for(int jk = 0; jk < 4; jk++){
                for(int q = 0; q < 32; q++){
                    for(int k = 0; k < 32; k++){
                        char bit = (sharedA[q * jk * TILE_WIDTH + tid + k]) & 1;
                        printf("%u,", bit);
                    }
                    printf("\n");
                }
            }/**/
            printf("B: i = %d\n", i);
            for(int jk = 0; jk < 4; jk++){
                for(int q = 0; q < 32; q++){
                    for(int k = 0; k < 32; k++){
                        char bit = (sharedB[q * jk * TILE_WIDTH + tid + k]) & 1;
                        printf("%u,", bit);
                    }
                    printf("\n");
                }
            }/**/
            
            /*printf("transposeB 0 through 3: ");
            for(int k = 0; k < 4; k++){
                for(int j = 0; j < 8; j++){
                    char bit = (transposeB[tid + k] >> (7 - j)) & 1;
                    printf("%u", bit);
                }
                printf(" ");
            }
            printf("\n");/**/
        }
        __syncthreads();
        /*for(int j = 0; j < TILE_WIDTH; j++){
            //pValueFloat[0] ^= (sharedFloatA[threadIdx.y * TILE_WIDTH + j]) & (sharedFloatB[j * TILE_WIDTH + threadIdx.x]);
            //pValueFloat[0] ^= (sharedFloatA[threadIdx.y * TILE_WIDTH + j]) & (transposeFloatB[threadIdx.x * TILE_WIDTH + j]);
            pValueFloat ^= sharedFloatA[threadIdx.y * TILE_WIDTH + j] & sharedFloatB[threadIdx.x * TILE_WIDTH + j];
        }/**/
        /*if(blockIdx.x == 0 && blockIdx.y == 0 && tid == 0){
            uint32_t temp = 0;
            for(int q = 0; q < 32; q++){
                temp ^= sharedFloatA[q] & sharedFloatB[q];
            }
            for(int j = 0; j < 32; j++){
                char bit = (temp >> (31 - j)) & 1;
                printf("%u", bit);
            }
            printf("\n");
        }/**/
        /*for(int q = 0; q < 4; q++){
            for(int j = 0; j < TILE_WIDTH; j++){
                pValue[q] ^= (sharedA[threadIdx.y * TILE_WIDTH + j] & sharedB[j * TILE_WIDTH + threadIdx.x]);
            }
        }/**/
        /*for(int j = 0; j < TILE_WIDTH; j++){
                shortValue ^= (sharedA[threadIdx.y * TILE_WIDTH + j] & sharedB[j * TILE_WIDTH + threadIdx.x]);
        }/**/
        /*if(blockIdx.x == 0 && blockIdx.y == 0 && tid == 0){// && i == ((colA - 1)/(TILE_WIDTH * 4))){
        printf("pValueFloat: ");
            for(int j = 0; j < 32; j++){
                char bit = (pValueFloat >> (31 - j)) & 1;
                printf("%u", bit);
            }
            printf("\n");
        }/**/
        __syncthreads();
    }
    /*if(blockIdx.x == 0 && blockIdx.y == 0 && tid == 0){
        printf("pValueFloat: ");
            for(int j = 0; j < 32; j++){
                char bit = (pValueFloat >> (31 - j)) & 1;
                printf("%u", bit);
            }
            printf("\n");
    }/**/
    /*for(int i = 0; i < 4; i++){
        //pValue[0] ^= pValue[i];
        //shortValue ^= pValue[i] & 1;
        shortValue ^= pValueFloat & 1;
        pValueFloat >>= 8;
    }/**/
    /*for(int i = 1; i < 4; i++){
        pValue[0] ^= pValue[i] & 1;
    }/**/
    //C[Row * colB + Col] = shortValue;
    /*if(Row < rowA && Col < colB){
        C[Row * colB + Col] = shortValue;
    }/**/
    //}
}/**/

/*__global__ void mult_kernel_compressed_data(HAMC_DATA_TYPE_t *A, HAMC_DATA_TYPE_t *B, HAMC_DATA_TYPE_t *C, int rowA, int rowB, int colA, int colB, int TILE_WIDTH)
{
    extern __shared__ HAMC_DATA_TYPE_t sharedArray[];
    
    HAMC_DATA_TYPE_t *sharedA = sharedArray;
    uint32_t *sharedFloatA = (uint32_t *)sharedA;
    uint32_t *sharedFloatB = &sharedFloatA[TILE_WIDTH * TILE_WIDTH];
    HAMC_DATA_TYPE_t *sharedB = (HAMC_DATA_TYPE_t *)sharedFloatB;
    
    uint8_t tempB[4];
    uint32_t *tempFloatB = (uint32_t *)tempB;
    
    uint8_t boundaryA[4];
    uint32_t *boundaryFloatA = (uint32_t *)boundaryA;
    uint8_t boundaryB[4];
    uint32_t *boundaryFloatB = (uint32_t *)boundaryB;
    
    uint32_t *floatA = (uint32_t *)A;
    uint32_t *floatB = (uint32_t *)B;
    
    int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    int tid = threadIdx.y * TILE_WIDTH + threadIdx.x;
    int tilePos = 0;

    uint32_t pValue = 0;
    HAMC_DATA_TYPE_t shortValue = 0;
    
    for(int i = 0; i < ((colA - 1)/(TILE_WIDTH / 4)) + 1; i++){
        tilePos = i * TILE_WIDTH;
        if((Row < rowA) && (tilePos + threadIdx.x < colA / 4)){
            sharedFloatA[tid] = floatA[Row * colA + tilePos + threadIdx.x];
        }
        else if(Row >= rowA){
            sharedFloatA[tid] = 0;
        }
        else{
            boundaryFloatA[0] = floatA[Row * colA + tilePos + threadIdx.x];
            for(int j = 0; j < colA % 4; j++){
                boundaryA[3 - j] &= (uint8_t)0;
            }
            sharedFloatA[tid] = boundaryFloatA[0];
        }
        if((Col < colB) && (tilePos + threadIdx.y < rowB)){
            tempFloatB[0] = floatB[((threadIdx.x / 8) + ((threadIdx.y + tilePos) * 4)) * colB + (blockIdx.x * TILE_WIDTH / 8) + (threadIdx.x % 8)];
        }
        else if((tilePos + threadIdx.y) * 4 >= rowB){
            tempFloatB[0] = 0;
        }
        else{
            boundaryFloatB[0] = floatB[((threadIdx.x / 8) + ((threadIdx.y + tilePos) * 4)) * colB + (blockIdx.x * TILE_WIDTH / 8) + (threadIdx.x % 8)];
            for(int j = 0; j < colB % 4; j++){
                boundaryB[3 - j] &= (uint8_t)0;
            }
            tempFloatB[0] = boundaryFloatB[0];
        }
        #pragma unroll
        for(int j = 0; j < 4; j++){
            sharedB[(j + threadIdx.y) * TILE_WIDTH + threadIdx.x] = tempB[j];
            //sharedB[tid * 4 + j] = B[(j + ((tilePos + threadIdx.y) * 4)) * colB + Col];
        }
        __syncthreads();
        for(int j = 0; j < TILE_WIDTH; j++){
            pValue ^= sharedFloatA[threadIdx.y * TILE_WIDTH + j] & sharedFloatB[j * TILE_WIDTH + threadIdx.x];
        }
        __syncthreads();
    }
    #pragma unroll
    for(int i = 0; i < 4; i++){
        shortValue ^= pValue & 1;
        pValue >>= 8;
    }
    __syncthreads();
    C[Row * colB + Col] = shortValue;
}/**/

__global__ void mult_kernel(HAMC_DATA_TYPE_t *A, HAMC_DATA_TYPE_t *B, HAMC_DATA_TYPE_t *C, int rowA, int rowB, int colA, int colB, int TILE_WIDTH)
{
    extern __shared__ HAMC_DATA_TYPE_t sharedArray[];
    
    //int TILE_WIDTH = (sizeof(sharedArray) / sizeof(sharedArray[0])) / 4;
    
    HAMC_DATA_TYPE_t *sharedA = sharedArray;
    HAMC_DATA_TYPE_t *sharedB = &sharedA[TILE_WIDTH * TILE_WIDTH];
    //extern __shared__ HAMC_DATA_TYPE_t sharedA[];
    //extern __shared__ HAMC_DATA_TYPE_t sharedB[];
    
    int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    int tid = threadIdx.y * TILE_WIDTH + threadIdx.x;
    int tilePos = 0;

    HAMC_DATA_TYPE_t pValue = 0;
  
    for(int i = 0; (i < ((colA - 1)/TILE_WIDTH) + 1) && (i < ((rowB - 1)/TILE_WIDTH) + 1); i++){
        tilePos = i * TILE_WIDTH;
        if((Row < rowA) && (tilePos + threadIdx.x < colA)){
            sharedA[tid] = A[Row * colA + tilePos + threadIdx.x];
        }
        else{
            sharedA[tid] = 0;
        }
        if((Col < colB) && (tilePos + threadIdx.y < rowB)){
            sharedB[tid] = B[(tilePos + threadIdx.y) * colB + Col];
        }
        else{
            sharedB[tid] = 0;
        }
        __syncthreads();
        
        if((Row < rowA) && (Col < colB)){
            for(int j = 0; j < TILE_WIDTH; j++){
                pValue ^= (sharedA[threadIdx.y * TILE_WIDTH + j] & sharedB[j * TILE_WIDTH + threadIdx.x]);
            }
        }
        
        __syncthreads();
    }/**/
    /*for(int i = 0; (i < ((colA - 1)/TILE_WIDTH) + 1); i++){
        tilePos = i * TILE_WIDTH;
        if((Row < rowA) && (tilePos + threadIdx.x < colA)){
            sharedA[tid] = A[Row * colA + tilePos + threadIdx.x];
        }
        else{
            sharedA[tid] = 0;
        }
        for(int k = 0; k < ((rowB - 1)/TILE_WIDTH) + 1; k++){
            if((Col < colB) && (tilePos + threadIdx.y < rowB)){
                sharedB[tid] = B[(tilePos + threadIdx.y) * colB + Col];
            }
            else{
                sharedB[tid] = 0;
            }
            __syncthreads();
            
            if((Row < rowA) && (Col < colB)){
                for(int j = 0; j < TILE_WIDTH; j++){
                    pValue ^= (sharedA[threadIdx.y * TILE_WIDTH + j] & sharedB[j * TILE_WIDTH + threadIdx.x]);
                }
            }
            
            __syncthreads();
        }
    }/**/
    if((Row < rowA) && (Col < colB)){
        C[Row * colB + Col] = pValue;
    }
}

__global__ void mult_kernel_debug(HAMC_DATA_TYPE_t *A, HAMC_DATA_TYPE_t *B, HAMC_DATA_TYPE_t *C, int rowA, int rowB, int colA, int colB, int TILE_WIDTH)
{
    extern __shared__ HAMC_DATA_TYPE_t sharedArray[];
    
    //int TILE_WIDTH = (sizeof(sharedArray) / sizeof(sharedArray[0])) / 4;
    
    HAMC_DATA_TYPE_t *sharedA = sharedArray;
    HAMC_DATA_TYPE_t *sharedB = &sharedA[TILE_WIDTH * TILE_WIDTH];
    //extern __shared__ HAMC_DATA_TYPE_t sharedA[];
    //extern __shared__ HAMC_DATA_TYPE_t sharedB[];
    
    int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    int tid = threadIdx.y * TILE_WIDTH + threadIdx.x;
    int tilePos = 0;

    HAMC_DATA_TYPE_t pValue = 0;
  
    for(int i = 0; (i < ((colA - 1)/TILE_WIDTH) + 1) && (i < ((rowB - 1)/TILE_WIDTH) + 1); i++){
        tilePos = i * TILE_WIDTH;
        if((Row < rowA) && (tilePos + threadIdx.x < colA)){
            sharedA[tid] = A[Row * colA + tilePos + threadIdx.x];
        }
        else{
            sharedA[tid] = 0;
        }
        if((Col < colB) && (tilePos + threadIdx.y < rowB)){
            sharedB[tid] = B[(tilePos + threadIdx.y) * colB + Col];
        }
        else{
            sharedB[tid] = 0;
        }
        __syncthreads();
        
        if(blockIdx.x == 0 && blockIdx.y == 0 && tid == 0){// && i == 0){
            printf("A: i = %d\n", i);
            for(int q = 0; q < 32; q++){
                for(int k = 0; k < 32; k++){
                    char bit = (sharedA[q * TILE_WIDTH + tid + k]) & 1;
                    printf("%u,", bit);
                }
                printf("\n");
            }
            printf("B: i = %d\n", i);
            for(int q = 0; q < 32; q++){
                for(int k = 0; k < 32; k++){
                    char bit = (sharedB[q * TILE_WIDTH + tid + k]) & 1;
                    printf("%u,", bit);
                }
                printf("\n");
            }
            
        }
        
        if((Row < rowA) && (Col < colB)){
            for(int j = 0; j < TILE_WIDTH; j++){
                pValue ^= (sharedA[threadIdx.y * TILE_WIDTH + j] & sharedB[j * TILE_WIDTH + threadIdx.x]);
            }
        }
        
        __syncthreads();
    }/**/
    if((Row < rowA) && (Col < colB)){
        C[Row * colB + Col] = pValue;
    }
}

/*__global__ void mult_kernel_async(HAMC_DATA_TYPE_t *A, HAMC_DATA_TYPE_t *B, HAMC_DATA_TYPE_t *C, int rowA, int rowB, int colA, int colB, int TILE_WIDTH)
{
    extern __shared__ HAMC_DATA_TYPE_t sharedArray[];
    
    //int TILE_WIDTH = (sizeof(sharedArray) / sizeof(sharedArray[0])) / 4;
    
    HAMC_DATA_TYPE_t *sharedA = sharedArray;
    HAMC_DATA_TYPE_t *sharedB = &sharedA[TILE_WIDTH * TILE_WIDTH];
    //extern __shared__ HAMC_DATA_TYPE_t sharedA[];
    //extern __shared__ HAMC_DATA_TYPE_t sharedB[];
    
    int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    int tid = threadIdx.y * TILE_WIDTH + threadIdx.x;
    int tilePos = 0;

    HAMC_DATA_TYPE_t pValue = 0;
  
    for(int i = 0; (i < ((colA - 1)/TILE_WIDTH) + 1) && (i < ((rowB - 1)/TILE_WIDTH) + 1); i++){
        tilePos = i * TILE_WIDTH;
        if((Row < rowA) && (tilePos + threadIdx.x < colA)){
            //sharedA[tid] = A[Row * colA + tilePos + threadIdx.x];
            __pipeline_memcpy_asyc(&sharedA[tid], &A[Row * colA + tilePos + threadIdx.x], sizeof(HAMC_DATA_TYPE_t));
        }
        else{
            sharedA[tid] = 0;
        }
        if((Col < colB) && (tilePos + threadIdx.y < rowB)){
            //sharedB[tid] = B[(tilePos + threadIdx.y) * colB + Col];
            __pipeline_memcpy_asyc(&sharedB[tid], &B[(tilePos + threadIdx.y) * colB + Col], sizeof(HAMC_DATA_TYPE_t));
        }
        else{
            sharedB[tid] = 0;
        }
        __pipeline_commit();
        __pipeline_wait_prior(0);

        __syncthreads();
        
        if((Row < rowA) && (Col < colB)){
            for(int j = 0; j < TILE_WIDTH; j++){
                pValue ^= (sharedA[threadIdx.y * TILE_WIDTH + j] & sharedB[j * TILE_WIDTH + threadIdx.x]);
            }
        }
        
        __syncthreads();
    }
    if((Row < rowA) && (Col < colB)){
        C[Row * colB + Col] = pValue;
    }
}*/

bin_matrix run_mult_kernel(bin_matrix A, bin_matrix B)
{
    int TILE_WIDTH = 32;
    
    if (A->cols != B->rows){
        printf("Matrices are incompatible, check dimensions...\n");
        exit(0);
    }

    HAMC_DATA_TYPE_t *deviceA;
    HAMC_DATA_TYPE_t *deviceB;
    HAMC_DATA_TYPE_t *deviceC;
    
    bin_matrix C = mat_init_cpu(A->rows, B->cols);
    
    hipMalloc((void **) &deviceA, A->cols * A->rows * sizeof(HAMC_DATA_TYPE_t));
    hipMalloc((void **) &deviceB, B->cols * B->rows * sizeof(HAMC_DATA_TYPE_t));
    hipMalloc((void **) &deviceC, B->cols * A->rows * sizeof(HAMC_DATA_TYPE_t));
    
    hipMemcpy(deviceA, A->data, A->cols * A->rows * sizeof(HAMC_DATA_TYPE_t), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, B->data, B->cols * B->rows * sizeof(HAMC_DATA_TYPE_t), hipMemcpyHostToDevice);
    
    dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    int x_blocks = ((B->cols - 1)/TILE_WIDTH) + 1;
    int y_blocks = ((A->rows - 1)/TILE_WIDTH) + 1;
    dim3 DimGrid(x_blocks, y_blocks, 1);
    
    mult_kernel<<<DimGrid, DimBlock, 2 * TILE_WIDTH * TILE_WIDTH * sizeof(HAMC_DATA_TYPE_t)>>>(deviceA, deviceB, deviceC, A->rows, B->rows, A->cols, B->cols, TILE_WIDTH);
    
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
    
    hipMemcpy(C->data, deviceC, C->cols * C->rows * sizeof(HAMC_DATA_TYPE_t), hipMemcpyDeviceToHost);
    
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
    //printf("C Row: %d, C Col: %d\n", C->rows)
    return C;
}

bin_matrix run_mult_kernel(bin_matrix A, bin_matrix B, int TILE_WIDTH)
{
    //int TILE_WIDTH = tile_width;
    
    if (A->cols != B->rows){
        printf("Matrices are incompatible, check dimensions...\n");
        exit(0);
    }

    HAMC_DATA_TYPE_t *deviceA;
    HAMC_DATA_TYPE_t *deviceB;
    HAMC_DATA_TYPE_t *deviceC;
    
    bin_matrix C = mat_init_cpu(A->rows, B->cols);
    
    hipMalloc((void **) &deviceA, A->cols * A->rows * sizeof(HAMC_DATA_TYPE_t));
    hipMalloc((void **) &deviceB, B->cols * B->rows * sizeof(HAMC_DATA_TYPE_t));
    hipMalloc((void **) &deviceC, B->cols * A->rows * sizeof(HAMC_DATA_TYPE_t));
    
    hipMemcpy(deviceA, A->data, A->cols * A->rows * sizeof(HAMC_DATA_TYPE_t), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, B->data, B->cols * B->rows * sizeof(HAMC_DATA_TYPE_t), hipMemcpyHostToDevice);
    
    dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    int x_blocks = ((B->cols - 1)/TILE_WIDTH) + 1;
    int y_blocks = ((A->rows - 1)/TILE_WIDTH) + 1;
    dim3 DimGrid(x_blocks, y_blocks, 1);
    
    mult_kernel<<<DimGrid, DimBlock, 2 * TILE_WIDTH * TILE_WIDTH * sizeof(HAMC_DATA_TYPE_t)>>>(deviceA, deviceB, deviceC, A->rows, B->rows, A->cols, B->cols, TILE_WIDTH);
    
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
    
    hipMemcpy(C->data, deviceC, C->cols * C->rows * sizeof(HAMC_DATA_TYPE_t), hipMemcpyDeviceToHost);
    
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    return C;
}

bin_matrix run_mult_kernel_debug(bin_matrix A, bin_matrix B, int TILE_WIDTH)
{
    //int TILE_WIDTH = tile_width;
    
    if (A->cols != B->rows){
        printf("Matrices are incompatible, check dimensions...\n");
        exit(0);
    }

    HAMC_DATA_TYPE_t *deviceA;
    HAMC_DATA_TYPE_t *deviceB;
    HAMC_DATA_TYPE_t *deviceC;
    
    bin_matrix C = mat_init_cpu(A->rows, B->cols);
    
    hipMalloc((void **) &deviceA, A->cols * A->rows * sizeof(HAMC_DATA_TYPE_t));
    hipMalloc((void **) &deviceB, B->cols * B->rows * sizeof(HAMC_DATA_TYPE_t));
    hipMalloc((void **) &deviceC, B->cols * A->rows * sizeof(HAMC_DATA_TYPE_t));
    
    hipMemcpy(deviceA, A->data, A->cols * A->rows * sizeof(HAMC_DATA_TYPE_t), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, B->data, B->cols * B->rows * sizeof(HAMC_DATA_TYPE_t), hipMemcpyHostToDevice);
    
    dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    int x_blocks = ((B->cols - 1)/TILE_WIDTH) + 1;
    int y_blocks = ((A->rows - 1)/TILE_WIDTH) + 1;
    dim3 DimGrid(x_blocks, y_blocks, 1);
    
    mult_kernel_debug<<<DimGrid, DimBlock, 2 * TILE_WIDTH * TILE_WIDTH * sizeof(HAMC_DATA_TYPE_t)>>>(deviceA, deviceB, deviceC, A->rows, B->rows, A->cols, B->cols, TILE_WIDTH);
    
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
    
    hipMemcpy(C->data, deviceC, C->cols * C->rows * sizeof(HAMC_DATA_TYPE_t), hipMemcpyDeviceToHost);
    
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    return C;
}

bin_matrix run_mult_kernel_test(bin_matrix A, bin_matrix B, int TILE_WIDTH)
{
    //int TILE_WIDTH = tile_width;
    
    if (A->cols != B->rows){
        printf("Matrices are incompatible, check dimensions...\n");
        exit(0);
    }

    HAMC_DATA_TYPE_t *deviceA;
    HAMC_DATA_TYPE_t *deviceB;
    HAMC_DATA_TYPE_t *deviceC;
    
    bin_matrix C = mat_init_cpu(A->rows, B->cols);
    
    hipMalloc((void **) &deviceA, A->cols * A->rows * sizeof(HAMC_DATA_TYPE_t));
    hipMalloc((void **) &deviceB, B->cols * B->rows * sizeof(HAMC_DATA_TYPE_t));
    hipMalloc((void **) &deviceC, B->cols * A->rows * sizeof(HAMC_DATA_TYPE_t));
    
    hipMemcpy(deviceA, A->data, A->cols * A->rows * sizeof(HAMC_DATA_TYPE_t), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, B->data, B->cols * B->rows * sizeof(HAMC_DATA_TYPE_t), hipMemcpyHostToDevice);
    
    dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    int x_blocks = ((B->cols - 1)/TILE_WIDTH) + 1;
    int y_blocks = ((A->rows - 1)/TILE_WIDTH) + 1;
    dim3 DimGrid(x_blocks, y_blocks, 1);
    
    mult_kernel_compressed_data<<<DimGrid, DimBlock, 2 * TILE_WIDTH * TILE_WIDTH * sizeof(float)>>>(deviceA, deviceB, deviceC, A->rows, B->rows, A->cols, B->cols, TILE_WIDTH);
    //mult_kernel_compressed_data<<<DimGrid, DimBlock, 2 * TILE_WIDTH * TILE_WIDTH * sizeof(float)>>>(deviceA, deviceB, deviceC, A->rows, B->rows, A->cols, B->cols, TILE_WIDTH);
    //mult_kernel_outer_product<<<DimGrid, DimBlock, TILE_WIDTH * TILE_WIDTH * sizeof(HAMC_DATA_TYPE_t)>>>(deviceA, deviceB, deviceC, A->rows, B->rows, A->cols, B->cols, TILE_WIDTH);
    
    hipDeviceSynchronize();
    /*hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
    */
    hipMemcpy(C->data, deviceC, C->cols * C->rows * sizeof(HAMC_DATA_TYPE_t), hipMemcpyDeviceToHost);
    
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    return C;
}

#endif /* HAMC_MULTIPLY_MATRIX_H */
