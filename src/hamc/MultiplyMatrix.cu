#include "hip/hip_runtime.h"

#ifndef HAMC_MULTIPLY_MATRIX_H
#define HAMC_MULTIPLY_MATRIX_H

#include <stdio.h>
#include <hip/hip_runtime.h>
//#include <cuda/pipeline>
#include "hamc_common.h"

//#define TILE_WIDTH 16

//int TILE_WIDTH = 16;

/*__global__ void mult_kernel_outer_product(HAMC_DATA_TYPE_t *A, HAMC_DATA_TYPE_t *B, HAMC_DATA_TYPE_t *C, int rowA, int rowB, int colA, int colB, int TILE_WIDTH)
{
    extern __shared__ HAMC_DATA_TYPE_t sharedA[];
    
    //int TILE_WIDTH = (sizeof(sharedArray) / sizeof(sharedArray[0])) / 4;
    
    //HAMC_DATA_TYPE_t *sharedA = sharedArray;
    //HAMC_DATA_TYPE_t *sharedB = &sharedA[TILE_WIDTH * TILE_WIDTH];
    //extern __shared__ HAMC_DATA_TYPE_t sharedA[];
    //extern __shared__ HAMC_DATA_TYPE_t sharedB[];
    
    int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    int tid = threadIdx.y * TILE_WIDTH + threadIdx.x;
    int tilePos = 0;

    HAMC_DATA_TYPE_t b = 0;
    HAMC_DATA_TYPE_t pValue = 0;
  
    for(int i = 0; (i < ((colA - 1)/TILE_WIDTH) + 1) && (i < ((rowB - 1)/TILE_WIDTH) + 1); i++){
        tilePos = i * TILE_WIDTH;
        if((Row < rowA) && (tilePos + threadIdx.x < colA)){
            sharedA[tid] = A[Row * colA + tilePos + threadIdx.x];
        }
        else{
            sharedA[tid] = 0;
        }
        if((Col < colB) && (tilePos + threadIdx.y < rowB)){
            b = B[(tilePos + threadIdx.y) * colB + Col];
        }
        else{
            b = 0;
        }
        __syncthreads();
        
        if((Row < rowA) && (Col < colB)){
            for(int j = 0; j < TILE_WIDTH; j++){
                b = B[(j * colB) +  * (blockDim.x * blockDim.y)];
                pValue ^= (sharedA[threadIdx.y * TILE_WIDTH + j] & b);
            }
        }
        
        __syncthreads();
    }
    if((Row < rowA) && (Col < colB)){
        C[Row * colB + Col] = pValue;
    }
}/**/

/*__global__ void mult_kernel_register_blocked(HAMC_DATA_TYPE_t *A, HAMC_DATA_TYPE_t *B, HAMC_DATA_TYPE_t *C, int rowA, int rowB, int colA, int colB){
    __shared__ HAMC_DATA_TYPE_t sharedA[64*64];
    __shared__ HAMC_DATA_TYPE_t sharedB[64*64];
    
    int tile = 64;
    HAMC_DATA_TYPE_t regC[16];
    
    int tileRow = blockIdx.y * tile;
    int tileCol = blockIdx.x * tile;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;//0-255
    int tilePosX = tid % tile;//0-63
    int tilePosY = tid / tile;//0-3
    int row = 0, sharedIndex = 0;
    int stride = (blockDim.x * blockDim.y) / tile;
    for(int i = 0; (i < ((colA - 1)/tile) + 1) && (i < ((rowB - 1)/tile) + 1) ; i++){
        for(int j = 0; j < (tile * tile) / (blockDim.x * blockDim.y); j++){//0-15
            row = j * stride + tilePosY;
            sharedIndex = row * tile + tilePosX;
            if(((tileRow + row) < rowA) && (((i * tile) + tilePosX) < colA)){
                sharedA[sharedIndex] = A[((tileRow + row) * colA) + (i * tile) + tilePosX];
            }
            else{
                sharedA[sharedIndex] = 0;
            }
            if(((row + (i * tile)) < rowB) && ((tileCol + tilePosX) < colB)){
                sharedB[sharedIndex] = B[(colB * (row + (i * tile))) + tileCol + tilePosX];
            }
            else{
                sharedB[sharedIndex] = 0;
            }
        }
        __syncthreads();
        for(int j = 0; j < blockDim.x; j++){
            //stride = j * blockDim.x / tile;
            for(int k = 0; k < tile; k++){
                regC[j] ^= sharedA[((j * blockDim.x / tile) + (tid % blockDim.x)) * tile + k] & sharedB[(k * tile) + (j * blockDim.x / tile) + (tid % blockDim.x)];
            }
        }
        __syncthreads();
    }
    __syncthreads();
    for(int i = 0; i < blockDim.x; i++){
        C[(tileRow + tid % blockDim.x + blockDim.y * (i / stride) * colB) + tileCol + tid % tile] = regC[i];
    }
}/**/

__global__ void mult_kernel_compressed_data(HAMC_DATA_TYPE_t *A, HAMC_DATA_TYPE_t *B, HAMC_DATA_TYPE_t *C, int rowA, int rowB, int colA, int colB, int TILE_WIDTH)
{
    extern __shared__ HAMC_DATA_TYPE_t sharedArray[];
    
    HAMC_DATA_TYPE_t *sharedA = sharedArray;
    uint32_t *sharedFloatA = (uint32_t *)sharedA;
    HAMC_DATA_TYPE_t *sharedB = &sharedA[TILE_WIDTH * TILE_WIDTH];
    uint32_t *sharedFloatB = (uint32_t *)sharedB;
    
    uint32_t *floatA = (uint32_t *)A;
    
    int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    int tid = threadIdx.y * TILE_WIDTH + threadIdx.x;
    int tilePos = 0;

    uint32_t pValue = 0;
    HAMC_DATA_TYPE_t shortValue = 0;
    
    for(int i = 0; i < ((colA - 1)/(TILE_WIDTH / 4)) + 1; i++){
        tilePos = i * TILE_WIDTH;
        sharedFloatA[tid] = floatA[Row * colA + tilePos + threadIdx.x];
        for(int j = 0; j < 4; j++){
            sharedB[tid * 4 + j] = B[(j + ((tilePos + threadIdx.y) * 4)) * colB + Col];
        }
        __syncthreads();
        for(int j = 0; j < TILE_WIDTH; j++){
            pValue ^= (uint32_t)sharedFloatA[threadIdx.y * TILE_WIDTH + j] & (uint32_t)sharedFloatB[j * TILE_WIDTH + threadIdx.x];
        }
    }
    //TODO: xor all pValue bits
    for(int i = 0; i < 4; i++){
        shortValue ^= pValue & 1;
        pValue >>= 8;
    }
    C[Row * colB + Col] = pValue;
}/**/

/*__global__ void mult_kernel_compressed_data(HAMC_DATA_TYPE_t *A, HAMC_DATA_TYPE_t *B, HAMC_DATA_TYPE_t *C, int rowA, int rowB, int colA, int colB, int TILE_WIDTH)
{
    extern __shared__ HAMC_DATA_TYPE_t sharedArray[];
    
    HAMC_DATA_TYPE_t *sharedA = sharedArray;
    uint32_t *sharedFloatA = (uint32_t *)sharedA;
    uint32_t *sharedFloatB = &sharedFloatA[TILE_WIDTH * TILE_WIDTH];
    HAMC_DATA_TYPE_t *sharedB = (HAMC_DATA_TYPE_t *)sharedFloatB;
    
    uint8_t tempB[4];
    uint32_t *tempFloatB = (uint32_t *)tempB;
    
    uint8_t boundaryA[4];
    uint32_t *boundaryFloatA = (uint32_t *)boundaryA;
    uint8_t boundaryB[4];
    uint32_t *boundaryFloatB = (uint32_t *)boundaryB;
    
    uint32_t *floatA = (uint32_t *)A;
    uint32_t *floatB = (uint32_t *)B;
    
    int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    int tid = threadIdx.y * TILE_WIDTH + threadIdx.x;
    int tilePos = 0;

    uint32_t pValue = 0;
    HAMC_DATA_TYPE_t shortValue = 0;
    
    for(int i = 0; i < ((colA - 1)/(TILE_WIDTH / 4)) + 1; i++){
        tilePos = i * TILE_WIDTH;
        if((Row < rowA) && (tilePos + threadIdx.x < colA / 4)){
            sharedFloatA[tid] = floatA[Row * colA + tilePos + threadIdx.x];
        }
        else if(Row >= rowA){
            sharedFloatA[tid] = 0;
        }
        else{
            boundaryFloatA[0] = floatA[Row * colA + tilePos + threadIdx.x];
            for(int j = 0; j < colA % 4; j++){
                boundaryA[3 - j] &= (uint8_t)0;
            }
            sharedFloatA[tid] = boundaryFloatA[0];
        }
        if((Col < colB) && (tilePos + threadIdx.y < rowB)){
            tempFloatB[0] = floatB[((threadIdx.x / 8) + ((threadIdx.y + tilePos) * 4)) * colB + (blockIdx.x * TILE_WIDTH / 8) + (threadIdx.x % 8)];
        }
        else if((tilePos + threadIdx.y) * 4 >= rowB){
            tempFloatB[0] = 0;
        }
        else{
            boundaryFloatB[0] = floatB[((threadIdx.x / 8) + ((threadIdx.y + tilePos) * 4)) * colB + (blockIdx.x * TILE_WIDTH / 8) + (threadIdx.x % 8)];
            for(int j = 0; j < colB % 4; j++){
                boundaryB[3 - j] &= (uint8_t)0;
            }
            tempFloatB[0] = boundaryFloatB[0];
        }
        #pragma unroll
        for(int j = 0; j < 4; j++){
            sharedB[(j + threadIdx.y) * TILE_WIDTH + threadIdx.x] = tempB[j];
            //sharedB[tid * 4 + j] = B[(j + ((tilePos + threadIdx.y) * 4)) * colB + Col];
        }
        __syncthreads();
        for(int j = 0; j < TILE_WIDTH; j++){
            pValue ^= sharedFloatA[threadIdx.y * TILE_WIDTH + j] & sharedFloatB[j * TILE_WIDTH + threadIdx.x];
        }
        __syncthreads();
    }
    #pragma unroll
    for(int i = 0; i < 4; i++){
        shortValue ^= pValue & 1;
        pValue >>= 8;
    }
    __syncthreads();
    C[Row * colB + Col] = shortValue;
}/**/

__global__ void mult_kernel(HAMC_DATA_TYPE_t *A, HAMC_DATA_TYPE_t *B, HAMC_DATA_TYPE_t *C, int rowA, int rowB, int colA, int colB, int TILE_WIDTH)
{
    extern __shared__ HAMC_DATA_TYPE_t sharedArray[];
    
    //int TILE_WIDTH = (sizeof(sharedArray) / sizeof(sharedArray[0])) / 4;
    
    HAMC_DATA_TYPE_t *sharedA = sharedArray;
    HAMC_DATA_TYPE_t *sharedB = &sharedA[TILE_WIDTH * TILE_WIDTH];
    //extern __shared__ HAMC_DATA_TYPE_t sharedA[];
    //extern __shared__ HAMC_DATA_TYPE_t sharedB[];
    
    int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    int tid = threadIdx.y * TILE_WIDTH + threadIdx.x;
    int tilePos = 0;

    HAMC_DATA_TYPE_t pValue = 0;
  
    for(int i = 0; (i < ((colA - 1)/TILE_WIDTH) + 1) && (i < ((rowB - 1)/TILE_WIDTH) + 1); i++){
        tilePos = i * TILE_WIDTH;
        if((Row < rowA) && (tilePos + threadIdx.x < colA)){
            sharedA[tid] = A[Row * colA + tilePos + threadIdx.x];
        }
        else{
            sharedA[tid] = 0;
        }
        if((Col < colB) && (tilePos + threadIdx.y < rowB)){
            sharedB[tid] = B[(tilePos + threadIdx.y) * colB + Col];
        }
        else{
            sharedB[tid] = 0;
        }
        __syncthreads();
        
        if((Row < rowA) && (Col < colB)){
            for(int j = 0; j < TILE_WIDTH; j++){
                pValue ^= (sharedA[threadIdx.y * TILE_WIDTH + j] & sharedB[j * TILE_WIDTH + threadIdx.x]);
            }
        }
        
        __syncthreads();
    }/**/
    /*for(int i = 0; (i < ((colA - 1)/TILE_WIDTH) + 1); i++){
        tilePos = i * TILE_WIDTH;
        if((Row < rowA) && (tilePos + threadIdx.x < colA)){
            sharedA[tid] = A[Row * colA + tilePos + threadIdx.x];
        }
        else{
            sharedA[tid] = 0;
        }
        for(int k = 0; k < ((rowB - 1)/TILE_WIDTH) + 1; k++){
            if((Col < colB) && (tilePos + threadIdx.y < rowB)){
                sharedB[tid] = B[(tilePos + threadIdx.y) * colB + Col];
            }
            else{
                sharedB[tid] = 0;
            }
            __syncthreads();
            
            if((Row < rowA) && (Col < colB)){
                for(int j = 0; j < TILE_WIDTH; j++){
                    pValue ^= (sharedA[threadIdx.y * TILE_WIDTH + j] & sharedB[j * TILE_WIDTH + threadIdx.x]);
                }
            }
            
            __syncthreads();
        }
    }/**/
    if((Row < rowA) && (Col < colB)){
        C[Row * colB + Col] = pValue;
    }
}

/*__global__ void mult_kernel_async(HAMC_DATA_TYPE_t *A, HAMC_DATA_TYPE_t *B, HAMC_DATA_TYPE_t *C, int rowA, int rowB, int colA, int colB, int TILE_WIDTH)
{
    extern __shared__ HAMC_DATA_TYPE_t sharedArray[];
    
    //int TILE_WIDTH = (sizeof(sharedArray) / sizeof(sharedArray[0])) / 4;
    
    HAMC_DATA_TYPE_t *sharedA = sharedArray;
    HAMC_DATA_TYPE_t *sharedB = &sharedA[TILE_WIDTH * TILE_WIDTH];
    //extern __shared__ HAMC_DATA_TYPE_t sharedA[];
    //extern __shared__ HAMC_DATA_TYPE_t sharedB[];
    
    int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    int tid = threadIdx.y * TILE_WIDTH + threadIdx.x;
    int tilePos = 0;

    HAMC_DATA_TYPE_t pValue = 0;
  
    for(int i = 0; (i < ((colA - 1)/TILE_WIDTH) + 1) && (i < ((rowB - 1)/TILE_WIDTH) + 1); i++){
        tilePos = i * TILE_WIDTH;
        if((Row < rowA) && (tilePos + threadIdx.x < colA)){
            //sharedA[tid] = A[Row * colA + tilePos + threadIdx.x];
            __pipeline_memcpy_asyc(&sharedA[tid], &A[Row * colA + tilePos + threadIdx.x], sizeof(HAMC_DATA_TYPE_t));
        }
        else{
            sharedA[tid] = 0;
        }
        if((Col < colB) && (tilePos + threadIdx.y < rowB)){
            //sharedB[tid] = B[(tilePos + threadIdx.y) * colB + Col];
            __pipeline_memcpy_asyc(&sharedB[tid], &B[(tilePos + threadIdx.y) * colB + Col], sizeof(HAMC_DATA_TYPE_t));
        }
        else{
            sharedB[tid] = 0;
        }
        __pipeline_commit();
        __pipeline_wait_prior(0);

        __syncthreads();
        
        if((Row < rowA) && (Col < colB)){
            for(int j = 0; j < TILE_WIDTH; j++){
                pValue ^= (sharedA[threadIdx.y * TILE_WIDTH + j] & sharedB[j * TILE_WIDTH + threadIdx.x]);
            }
        }
        
        __syncthreads();
    }
    if((Row < rowA) && (Col < colB)){
        C[Row * colB + Col] = pValue;
    }
}*/

bin_matrix run_mult_kernel(bin_matrix A, bin_matrix B)
{
    int TILE_WIDTH = 32;
    
    if (A->cols != B->rows){
        printf("Matrices are incompatible, check dimensions...\n");
        exit(0);
    }

    HAMC_DATA_TYPE_t *deviceA;
    HAMC_DATA_TYPE_t *deviceB;
    HAMC_DATA_TYPE_t *deviceC;
    
    bin_matrix C = mat_init_cpu(A->rows, B->cols);
    
    hipMalloc((void **) &deviceA, A->cols * A->rows * sizeof(HAMC_DATA_TYPE_t));
    hipMalloc((void **) &deviceB, B->cols * B->rows * sizeof(HAMC_DATA_TYPE_t));
    hipMalloc((void **) &deviceC, B->cols * A->rows * sizeof(HAMC_DATA_TYPE_t));
    
    hipMemcpy(deviceA, A->data, A->cols * A->rows * sizeof(HAMC_DATA_TYPE_t), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, B->data, B->cols * B->rows * sizeof(HAMC_DATA_TYPE_t), hipMemcpyHostToDevice);
    
    dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    int x_blocks = ((B->cols - 1)/TILE_WIDTH) + 1;
    int y_blocks = ((A->rows - 1)/TILE_WIDTH) + 1;
    dim3 DimGrid(x_blocks, y_blocks, 1);
    
    mult_kernel<<<DimGrid, DimBlock, 2 * TILE_WIDTH * TILE_WIDTH * sizeof(HAMC_DATA_TYPE_t)>>>(deviceA, deviceB, deviceC, A->rows, B->rows, A->cols, B->cols, TILE_WIDTH);
    
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
    
    hipMemcpy(C->data, deviceC, C->cols * C->rows * sizeof(HAMC_DATA_TYPE_t), hipMemcpyDeviceToHost);
    
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
    //printf("C Row: %d, C Col: %d\n", C->rows)
    return C;
}

bin_matrix run_mult_kernel(bin_matrix A, bin_matrix B, int TILE_WIDTH)
{
    //int TILE_WIDTH = tile_width;
    
    if (A->cols != B->rows){
        printf("Matrices are incompatible, check dimensions...\n");
        exit(0);
    }

    HAMC_DATA_TYPE_t *deviceA;
    HAMC_DATA_TYPE_t *deviceB;
    HAMC_DATA_TYPE_t *deviceC;
    
    bin_matrix C = mat_init_cpu(A->rows, B->cols);
    
    hipMalloc((void **) &deviceA, A->cols * A->rows * sizeof(HAMC_DATA_TYPE_t));
    hipMalloc((void **) &deviceB, B->cols * B->rows * sizeof(HAMC_DATA_TYPE_t));
    hipMalloc((void **) &deviceC, B->cols * A->rows * sizeof(HAMC_DATA_TYPE_t));
    
    hipMemcpy(deviceA, A->data, A->cols * A->rows * sizeof(HAMC_DATA_TYPE_t), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, B->data, B->cols * B->rows * sizeof(HAMC_DATA_TYPE_t), hipMemcpyHostToDevice);
    
    dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    int x_blocks = ((B->cols - 1)/TILE_WIDTH) + 1;
    int y_blocks = ((A->rows - 1)/TILE_WIDTH) + 1;
    dim3 DimGrid(x_blocks, y_blocks, 1);
    
    mult_kernel<<<DimGrid, DimBlock, 2 * TILE_WIDTH * TILE_WIDTH * sizeof(HAMC_DATA_TYPE_t)>>>(deviceA, deviceB, deviceC, A->rows, B->rows, A->cols, B->cols, TILE_WIDTH);
    
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
    
    hipMemcpy(C->data, deviceC, C->cols * C->rows * sizeof(HAMC_DATA_TYPE_t), hipMemcpyDeviceToHost);
    
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    return C;
}

bin_matrix run_mult_kernel_test(bin_matrix A, bin_matrix B, int TILE_WIDTH)
{
    //int TILE_WIDTH = tile_width;
    
    if (A->cols != B->rows){
        printf("Matrices are incompatible, check dimensions...\n");
        exit(0);
    }

    HAMC_DATA_TYPE_t *deviceA;
    HAMC_DATA_TYPE_t *deviceB;
    HAMC_DATA_TYPE_t *deviceC;
    
    bin_matrix C = mat_init_cpu(A->rows, B->cols);
    
    hipMalloc((void **) &deviceA, A->cols * A->rows * sizeof(HAMC_DATA_TYPE_t));
    hipMalloc((void **) &deviceB, B->cols * B->rows * sizeof(HAMC_DATA_TYPE_t));
    hipMalloc((void **) &deviceC, B->cols * A->rows * sizeof(HAMC_DATA_TYPE_t));
    
    hipMemcpy(deviceA, A->data, A->cols * A->rows * sizeof(HAMC_DATA_TYPE_t), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, B->data, B->cols * B->rows * sizeof(HAMC_DATA_TYPE_t), hipMemcpyHostToDevice);
    
    dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    int x_blocks = ((B->cols - 1)/TILE_WIDTH) + 1;
    int y_blocks = ((A->rows - 1)/TILE_WIDTH) + 1;
    dim3 DimGrid(x_blocks, y_blocks, 1);
    
    mult_kernel_compressed_data<<<DimGrid, DimBlock, 2 * TILE_WIDTH * TILE_WIDTH * sizeof(float)>>>(deviceA, deviceB, deviceC, A->rows, B->rows, A->cols, B->cols, TILE_WIDTH);
    //mult_kernel_outer_product<<<DimGrid, DimBlock, TILE_WIDTH * TILE_WIDTH * sizeof(HAMC_DATA_TYPE_t)>>>(deviceA, deviceB, deviceC, A->rows, B->rows, A->cols, B->cols, TILE_WIDTH);
    
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
    
    hipMemcpy(C->data, deviceC, C->cols * C->rows * sizeof(HAMC_DATA_TYPE_t), hipMemcpyDeviceToHost);
    
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    return C;
}

#endif /* HAMC_MULTIPLY_MATRIX_H */
