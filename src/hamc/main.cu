#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <wb.h>
#include <stdint.h>
#include <ctype.h>
#include <stdio.h>
#include <unistd.h>

#include "hamc_cpu_code.c"

#include "hamc_common.h"
#include "decrypt.cu"
#include "encrypt.cu"
#include "keygen.cu"
#include "hamc_e2e.cu"


#define CUDA_CHECK(ans) \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}

void printHelp();
void printWelcome()
{
    printf("HAMC Version %s0.1%s\n", YELLOW, NC);
    printf("Developed by Mitchell Dzurick, Mitchell Russel, James Kuban");
}


int main(int argc, char *argv[]) {
    printWelcome();

    /* variables for timing operations */

    /* input parameters */
    int n = 2, p = 500, w = 30, t = 10, seed = 10;
    char *outputFileName = NULL, *inputFileName = NULL, *action = NULL;

    char *keyFile = NULL;

    /* determines whether to run CPU based implementation, default no */
    bool cpu = false;

    bool verbose = false;

    int c;
    opterr = 0;
    while ((c = getopt (argc, argv, "a:n:p:w:t:i:o:hs:cs:vk:")) != -1)
        switch(c)
        {
            case 'k':
                keyFile = strdup(optarg);
                break;
            case 'v':
                verbose = true;
                break;
            case 'c':
                cpu = true;
                break;
            case 'n':
                n = atoi(optarg);
                break;
            case 's':
                seed = atoi(optarg);
                break;
            case 'p':
                p = atoi(optarg);
                break;
            case 'w':
                w = atoi(optarg);
                break;
            case 't':
                t = atoi(optarg);
                break;
            case 'i':
                inputFileName = strdup(optarg);
                break;
            case 'o':
                outputFileName = strdup(optarg);
                break;
            case 'a':
                action = strdup(optarg);
                break;
            case 'h':
                printHelp();
                return(1);
            default:
                abort();

        }

    int k = (n - 1) * p;

    bool test = false;
    if (!strcmp(action, (const char*)"test")) test = true;

    /* print input parameters */
    printf("\n");
    printf("Input Parameters:\n");
    if (!test) printf("\tInput file: %s%s%s\n", YELLOW, inputFileName, NC);
    if (!test) printf("\tOutput file: %s%s%s\n", YELLOW, outputFileName, NC);
    printf("\tGPU based execution: ");
    if (!cpu) printf("%son%s\n", GREEN, NC);
    else printf("%soff%s\n", RED, NC);
    printf("\tn: %s%d%s\n", YELLOW, n, NC);
    printf("\tp: %s%d%s\n", YELLOW, p, NC);
    printf("\tw: %s%d%s\n", YELLOW, w, NC);
    printf("\tt: %s%d%s\n", YELLOW, t, NC);
    printf("\tk: %s%d%s\n", YELLOW, k, NC);
    printf("\tseed: %s%d%s\n", YELLOW, seed, NC);
    printf("\taction: %s%s%s\n", YELLOW, action, NC);


    //TODO: make sure action is null-terminated before passing into strcmp
    if (!strcmp(action, (const char*)"keygen")) {
            run_keygen(n, p, t, w, seed, cpu, verbose);
    }
    else if (!strcmp(action, (const char*)"encrypt")) {
        run_encryption_from_key(inputFileName, keyFile, outputFileName, n, t, cpu, verbose);
    }
    else if (!strcmp(action, (const char*)"decrypt")) {
        //run_decrypt(inputFileName, outputFileName, n, p, t, w, seed, cpu, verbose);
    }
    else if (!strcmp(action, (const char*)"generate-message")) {
        generate_message(outputFileName, k);
    }
    else if (test) {
        test_hamc_e2e(n, p, t, w, seed, cpu, true);
    }
    else {
        printf("action %s not recognized\n", action);
    }
}


void printHelp(){
    printf("\n\nHAMC - Hardware Accelerated Mceliece Cryptosystem\n\n");

    printf("Run the program as such:\n");
    printf("  ./hamc [arguments]\n\n");

    printf("Available Arguments:\n");
    printf("[X] denotes that an argument is required\n");
    printf("\t-a [X] : actions: keygen encrypt decrypt test\n\n");
    printf("\t-c : Run CPU based execution\n\n");
    printf("\t-h : Print this help menu\n\n");
    printf("\t-i [X] : input filename\n\n");
    printf("\t-k [X] : key filename\n\n");
    printf("\t-n [X] : Weight of generator matrix rows \n\n");
    printf("\t-o [X] : output filename\n\n");
    printf("\t-p [X] : Size of matrix during key generation\n\n");
    printf("\t-s : Seed for random number generation\n\n");
    printf("\t-t [X] : Weight of Error Matrix rows\n\n");
    printf("\t-v : Verbose\n\n");
    printf("\t-w [X] : Weight of QC_MDPC code\n\n");

    printf("Example program execution:\n");
    printf("  ./hamc -a test -n 2 -p 1024 -t 10 -w 30 -s 10\n");
    printf("  ./hamc -a test -n 2 -p 500 -t 10 -w 30 -s 10\n");
    printf("  ./hamc -a test -n 2 -p 500 -t 10 -w 30 -s 10 -c\n");


}
