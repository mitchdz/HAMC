#ifndef ENCRYPT_KERNEL_H
#define ENCRYPT_KERNEL_H

#include <stdio.h>
#include <time.h>


#include "TransposeMatrix.cu"
#include "MultiplyMatrix.cu"
#include "hamc_cpu_code.c"

#define TILE_WIDTH_MULTIPLY 16

#ifndef HAMC_DATA_TYPE_t
#define HAMC_DATA_TYPE_t HAMC_DATA_TYPE_t
#endif

bin_matrix run_matrix_multiply_kernel(bin_matrix A, bin_matrix B)
{
    bin_matrix C = mat_init_cpu(A->rows, B->cols);

    /* allocate device memory */
    HAMC_DATA_TYPE_t *deviceA;
    HAMC_DATA_TYPE_t *deviceB;
    HAMC_DATA_TYPE_t *deviceC;
    hipMalloc((void **) &deviceA, A->rows * A->cols * sizeof(HAMC_DATA_TYPE_t));
    hipMalloc((void **) &deviceB, B->rows * B->cols * sizeof(HAMC_DATA_TYPE_t));
    hipMalloc((void **) &deviceC, C->rows * C->cols * sizeof(HAMC_DATA_TYPE_t));

    /* transfer host data to device */
    hipMemcpy(deviceA, A->data, A->rows * A->cols * sizeof(HAMC_DATA_TYPE_t), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, B->data, B->cols * B->rows * sizeof(HAMC_DATA_TYPE_t), hipMemcpyHostToDevice);

    printf("Starting multiply matrix kernel...\n");

     /* determine block and grid dimensions */
    /*dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    int x_blocks = ((A->rows - 1)/TILE_WIDTH) + 1;
    int y_blocks = ((A->cols - 1)/TILE_WIDTH) + 1;
    dim3 DimGrid(x_blocks, y_blocks, 1);

    mult_kernel<<<DimGrid, DimBlock>>> (deviceA, deviceB, deviceC, A->rows, B->rows, A->cols, B->cols);

    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));

    hipMemcpy(C->data, deviceC, C->rows * C->cols * sizeof(HAMC_DATA_TYPE_t), hipMemcpyDeviceToHost);*/

    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    return C;
}



void run_encryption_gpu(const char* inputFileName, const char* outputFileName,
        int n, int p, int t, int w, int seed)
{
    //TODO:
}

#endif /* ENCRYPT_KERNEL_H */
