#include "hip/hip_runtime.h"
#ifndef HAMC_DATA_TYPE_t
#define HAMC_DATA_TYPE_t HAMC_DATA_TYPE_t
#endif

#include "hamc_cpu_code.c"


#define mat_element_gpu(mat, cols, row_idx, col_idx) \
          mat[row_idx * (cols) + col_idx]


//TODO: make device function to generate NxN identity matrix



__global__ void binary_gaussian_elimination_with_pivot()
{
    // for each column k = 0 : n



}


// uses shared memory
// each thread handles a single column
__global__ void binary_inverse_square_matrix_naive(HAMC_DATA_TYPE_t *in, HAMC_DATA_TYPE_t *out, int rows, int cols)
{
    __shared__ HAMC_DATA_TYPE_t A[16];
    __shared__ HAMC_DATA_TYPE_t B[16];

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;

    if (idx == 0) {
        for (int i = 0; i < rows; i++) {
            for (int j = 0; j < cols; j++) {
                mat_element_gpu(A,cols,i,j) = mat_element_gpu(in,cols,i,j);

                if (i == j) B[i*rows  + j] = 1;
                else B[i*rows + j] = 0;
            }
        }
    }

    __syncthreads();

    if (idx == 0 ) {
        printf("KERNEL: rows: %d cols: %d\n", rows, cols);
        for (int i =0; i < rows; i++) {
            for (int j = 0; j < cols; j++) {
                printf("%hu ", mat_element_gpu(A, cols, i, j));
            }
            printf("\n");
        }
    }

    /* wait for identity matrix to be created */
    __syncthreads();


    printf("Finished setting up matrices\n");

    for(int i = 0; i < cols; i++) {
        printf("i: %d\n", i);
        __syncthreads();
        if(mat_element_gpu(A, cols, i, i) == 1) {
            __syncthreads();
            for(int j = 0; j <  rows; j++) {

                printf("\ti=%d ,j=%d ,A[j,i]=%d, \n", i, j, mat_element_gpu(A, cols, j, i));
                __syncthreads();
                if(i != j && mat_element_gpu(A, cols, j, i) == 1) {
                    printf("\t\t got into last if statement\n");
                    /* (i) notates the ith row of the matrix */
                    /* => denotes where the row which the output is stored in */

                    __syncthreads();

                    /* add rows for both A and B*/
                    //printf("i: %d j: %d\n", i, j );
                    for(int colid = 0; colid < cols; colid++) {
                        printf("\t\t\tcolid: %d from thread %d\n", colid, idx);
                        /* each thread handles adding a value in both A & B */
                        __syncthreads();
                        if (idx == colid) {

                            printf("\t\t\t\tidx: %d, colid: %d\n", idx, colid);

                            /* add rows to identity */
                            /* (i) ^ (j) => (j) */
                            //add_rows_new_cpu(B, i, j, 0, A->cols);
                            mat_element_gpu(B, cols, j, colid) =
                                (mat_element_gpu(B, cols, i, colid)
                                ^ mat_element_gpu(B, cols, j, colid));

                            /* A is special, we only XOR from i to cols */
                            if (colid >= i) {
                                /* add rows to input */
                                /* (i) ^ (j) => (j) from i to cols */
                                //add_rows_new_cpu(A, i, j, i, A->cols);
                                mat_element_gpu(A, cols, j, colid) =
                                    (mat_element_gpu(A, cols, i, colid)
                                    ^ mat_element_gpu(A, cols, j, colid));

                            }
                            __syncthreads();
                        }
                        __syncthreads();
                    }
                    __syncthreads();
                }
                __syncthreads();
            }
            __syncthreads();
        }
        else {
            __syncthreads();
            for(int k = i + 1; k < rows; k++) {
                printf("k: %d\n", k);
                __syncthreads();
                if(mat_element_gpu(A, cols, k, i) == 1) {
                    __syncthreads();
                    // for each column, XOR k and i, store into i
                    for(int colid = 0; colid < cols; colid++) {
                        /* each thread handles adding a val in both A & B */
                        __syncthreads();
                        if (idx == colid) {
                            /* add rows to identity */
                            /* (k) ^ (i) => (i) */
                            //add_rows_cpu(B, k, i);
                            mat_element_gpu(A, cols, k, i) =
                                (mat_element_gpu(A, cols, k, colid)
                                ^ mat_element_gpu(A, cols, i, colid));

                             /* add rows to input */
                             /* (k) ^ (i) => (i) */
                             //add_rows_cpu(A, k, i);
                             mat_element_gpu(B, cols, k, i) =
                                (mat_element_gpu(B, cols, k, colid)
                                ^ mat_element_gpu(B, cols, i, colid));
                        }
                        __syncthreads();

                    }
                    __syncthreads();
                    i = i - 1;
                    __syncthreads();
                    break;
                }
                __syncthreads();
            }
            __syncthreads();
        }
        __syncthreads();
    }

    __syncthreads();

    //if (idx == 0) {
    //    for (int i = 0; i < rows; i++) {
    //        for (int j = 0; j < rows; j++) {
    //            printf("%hu ", mat_element_gpu(B, cols, i, j));
    //        }
    //        printf("\n");
    //    }
    //}

    __syncthreads();

    //write to output
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < rows; j++) {
            mat_element_gpu(out, cols, i, j) = mat_element_gpu(B, cols, i, j);
        }
    }

    return;
}

bin_matrix run_inverse_kernel(bin_matrix A)
{
    bin_matrix C = mat_init_cpu(A->rows, A->cols);

    /* allocate device memory */
    HAMC_DATA_TYPE_t *deviceA;
    HAMC_DATA_TYPE_t *deviceB;
    hipMalloc((void **) &deviceA, A->rows * A->cols * sizeof(HAMC_DATA_TYPE_t));
    hipMalloc((void **) &deviceB, A->rows * A->cols * sizeof(HAMC_DATA_TYPE_t));

    /* transfer host data to device */
    hipMemcpy(deviceA, A->data, A->rows * A->cols * sizeof(HAMC_DATA_TYPE_t), hipMemcpyHostToDevice);

    printf("A from run_inverse_kernel:\n");
    for (int i =0; i < A->rows; i++) {
        for (int j = 0; j < A->cols; j++) {
            printf("%hu ", mat_element_gpu(A->data, A->cols, i, j));
        }
        printf("\n");
    }



    printf("Starting Inverse matrix kernel...\n");

    // /* determine block and grid dimensions */
    //dim3 DimBlock(TRANSPOSE_TILE_WIDTH, TRANSPOSE_TILE_WIDTH, 1);
    //int x_blocks = ((A->rows - 1)/TRANSPOSE_TILE_WIDTH) + 1;
    //int y_blocks = ((A->cols - 1)/TRANSPOSE_TILE_WIDTH) + 1;
    //dim3 DimGrid(x_blocks, y_blocks, 1);


    binary_inverse_square_matrix_naive<<<1, 4>>>
        (deviceA, deviceB, A->rows, A->cols);

    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));

    hipMemcpy(C->data, deviceB, A->rows * A->cols * sizeof(HAMC_DATA_TYPE_t), hipMemcpyDeviceToHost);

    hipFree(deviceA);
    hipFree(deviceB);

    return C;
}
