
#include <hip/hip_runtime.h>

#ifndef ushort
#define ushort unsigned short
#endif

#define TILE_DIM 32
#define BLOCK_ROWS 8

__global__ void transposeNaive(ushort *odata, const ushort *idata)
{
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
    odata[x*width + (y+j)] = idata[(y+j)*width + x];
}

__global__ void transposeCoalesced(ushort *odata, const ushort *idata)
{
  __shared__ float tile[TILE_DIM][TILE_DIM];

  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}

__global__ void TransposeSharedMem(ushort *odata, const ushort *idata)
{
  __shared__ float tile[TILE_DIM * TILE_DIM];

  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[(threadIdx.y+j)*TILE_DIM + threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*width + x] = tile[(threadIdx.y+j)*TILE_DIM + threadIdx.x];
}


__global__ void transpose(ushort *A, ushort *B, int rowA, int colA) {
	extern __shared__ ushort shared_B[];
	
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	
	if(row < rowA && col < colA){
		shared_B[col * rowA + row] = A[row * colA + col];
    }
	__syncthreads();
	
	if(row < rowA && col < colA){
		B[row * colA + col] = shared_B[row * colA + col];
    }
}
