#ifndef HAMC_SCRATCH_H
#define HAMC_SCRATCH_H

#include <wb.h>

#include <bits/getopt_core.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <stdint.h>

#include <sys/time.h>


#include "../../src/hamc/hamc_cpu_code.c"
#include "../../src/hamc/LU_inverse_plain.cu"



void run_find_forward_kernel(bin_matrix A)
{

    int numThreadsPerBlock = 1024;
    int numGrids = A->cols/numThreadsPerBlock + 1;

    dim3 dimgrid = dim3(numGrids);
    dim3 dimBlock = dim3(numThreadsPerBlock);


    HAMC_DATA_TYPE_t *deviceA;

    hipMalloc((void **)
        &deviceA, A->rows * A->cols * sizeof(HAMC_DATA_TYPE_t));


    HAMC_DATA_TYPE_t *deviceB;
    hipMalloc((void **)
        &deviceB, A->rows * A->cols * sizeof(HAMC_DATA_TYPE_t));

    hipMemcpy(deviceA, A->data, A->rows * A->cols * sizeof(HAMC_DATA_TYPE_t), 
        hipMemcpyHostToDevice);


    make_GF2_identity_gpu<<<1,1>>>(deviceB, A->rows);


    GF2_Forward_substitute<<<dimgrid, dimBlock>>> 
        (deviceA, deviceB, A->rows);

    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
         hipGetErrorString(cudaerr));

    return;
}



int main(int argc, char *argv[]){

    printf("Performing LU_inverse_forward_subsitution test\n");

    bool verbose = true;

    int n = 2;
    int p = 512;
    int t = 10;
    int w = 30;
    int seed = 10;


    int opt;
    while ((opt = getopt(argc, argv, "n:")) != -1){
        switch(opt){
            case 'n':
                p = atoi(optarg);
                break;
        }
    }

    bin_matrix invertible_matrix;

    mdpc code;

    code = qc_mdpc_init_cpu(n, p, t, w, seed);
    invertible_matrix = make_matrix_cpu(
        code->p, code->p, 
        splice_cpu(code->row, (code->n0 - 1) * code->p, code->n), 
        1);

    if (verbose) {
        printf("Input matrix size: %dx%d\n",
            invertible_matrix->rows, invertible_matrix->cols);
    }

    run_find_forward_kernel(invertible_matrix);


    if (verbose) printf("Freeing allocated memory...\n");
    if (invertible_matrix != NULL) free(invertible_matrix);

    return 0;
}


#endif /* HAMC_SCRATCH_H */