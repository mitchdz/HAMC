#ifndef HAMC_SCRATCH_H
#define HAMC_SCRATCH_H

#include <wb.h>

#include <bits/getopt_core.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <stdint.h>

#include <sys/time.h>


#include "../../src/hamc/hamc_cpu_code.c"
#include "../../src/hamc/LU_inverse_plain.cu"



void run_find_max_kernel(bin_matrix A)
{
    HAMC_DATA_TYPE_t *deviceA;

    hipMalloc((void **)
        &deviceA, A->rows * A->cols * sizeof(HAMC_DATA_TYPE_t));

    hipMemcpy(deviceA, A->data, A->rows * A->cols * sizeof(HAMC_DATA_TYPE_t), 
        hipMemcpyHostToDevice);

    int *deviceIPIV;
    hipMalloc((void **) &deviceIPIV, A->rows * sizeof(int));

    GF2_LU_decompose_find_max_row<<<1,1>>>(deviceA, deviceIPIV, 0, 0);

    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
         hipGetErrorString(cudaerr));

    return;
}


int main(int argc, char *argv[]){

    bool verbose = true;

    int n = 2;
    int p = 512;
    int t = 10;
    int w = 30;
    int seed = 10;

    int opt;
    while ((opt = getopt(argc, argv, "n:")) != -1){
        switch(opt){
            case 'n':
                p = atoi(optarg);
                break;
        }
    }

    bin_matrix invertible_matrix;

    mdpc code;

    code = qc_mdpc_init_cpu(n, p, t, w, seed);
    invertible_matrix = make_matrix_cpu(code->p, code->p, splice_cpu(code->row, (code->n0 - 1) * code->p, code->n), 1);

    if (verbose) {
        printf("Input matrix size: %dx%d\n",
            invertible_matrix->rows, invertible_matrix->cols);
    }

    run_find_max_kernel(invertible_matrix);


    if (verbose) printf("Freeing allocated memory...\n");
    if (invertible_matrix != NULL) free(invertible_matrix);

    return 0;
}


#endif /* HAMC_SCRATCH_H */