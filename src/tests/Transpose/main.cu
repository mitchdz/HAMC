#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <wb.h>
#include <stdint.h>
#include <ctype.h>
#include <stdio.h>
#include <unistd.h>
#include <iostream>
#include <time.h>

#include "../../hamc/hamc_cpu_code.c"
#include "../../hamc/hamc_common.h"
#include "../../hamc/TransposeMatrix.cu"

#define TILE_WIDTH 16
#define BLOCK_DIM 16
#define BLOCK_SIZE 16

#define TILE_DIM 16
#define BLOCK_ROWS 8


#define CUDA_CHECK(ans)                                                   \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess){
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

void printHelp()
{
    printf("run this executable with the following flags\n");
    printf("\n");
    printf("\t-a <input0 file name>\n");
    printf("\t-e <expected solution file name>\n");
    printf("\t-c \n");
    printf("\t  run CPU based execution\n");
}

static HAMC_DATA_TYPE_t *generate_data(int height, int width)
{
    HAMC_DATA_TYPE_t *data = (HAMC_DATA_TYPE_t *)malloc(sizeof(HAMC_DATA_TYPE_t) * width * height);
    int i;
    for (i = 0; i < width * height; i++) {
        data[i] = (HAMC_DATA_TYPE_t)(rand() % 2); // 0 or 1
    }
    return data;
}

void run_test(int x, int y, int type, int cpu)
{
    printf("matrix dim: %d %d\n",x, y);
    if (cpu)
        printf("type: cpu\n");
    else
        printf("type: %d\n", type);


    clock_t start, end;
    double cpu_time_used;

    HAMC_DATA_TYPE_t *raw_data = (HAMC_DATA_TYPE_t *)malloc(sizeof(HAMC_DATA_TYPE_t) * x * y);
    raw_data = generate_data(x, y);

    bin_matrix input = mat_init_cpu(x,y);
    input->data = raw_data;

    /* CPU execution time */
    start = clock();

    bin_matrix CPU_BIN = NULL;

    if (cpu) CPU_BIN = transpose_cpu(input);

    end = clock();
    cpu_time_used = ((double) (end - start))/ CLOCKS_PER_SEC;
    printf("CPU time: %lf\n", cpu_time_used);


    /* GPU execution time */
    start = clock();
    bin_matrix GPU_BIN = NULL;
    if (!cpu) {
        if (type == 1) 
            GPU_BIN = run_transpose_kernel(input);
        else if (type == 0) {
            GPU_BIN = run_transpose_kernel_naive(input);
        }
    }

    end = clock();
    cpu_time_used = ((double) (end - start))/ CLOCKS_PER_SEC;
    printf("GPU time: %lf\n", cpu_time_used);

    if (input) free(input);
    if (raw_data) free(raw_data);
    if (CPU_BIN) delete_bin_matrix(CPU_BIN);
    if (GPU_BIN) delete_bin_matrix(GPU_BIN);
}



int main(int argc, char *argv[])
{
    printf("Transpose matrix unit test\n");

    bin_matrix A;
    bin_matrix B;
    int numRowsA;
    int numColsA;
    int numRowsS;
    int numColsS;
    HAMC_DATA_TYPE_t *hostA;
    HAMC_DATA_TYPE_t *sol;
    char *input0 = NULL;
    char *expected = NULL;
    bool cpu_exec = false;
    bool solved = true;

    char *action = NULL;
    int x, y;

    int type = 1;


    int opt;
    while ((opt = getopt(argc, argv, "i:b:e:o:cx:y:a:t:")) != -1){
        switch(opt){
            case 'y':
                y = atoi(optarg);
                break;
             case 'x':
                x = atoi(optarg);
                break;
            case 'a':
                action = strdup(optarg);
                break;
            case 'i':
                input0 = strdup(optarg);
                break;
            case 'e':
                expected = strdup(optarg);
                break;
            case 'o':
                //input0 = strdup(optarg);
                break;
            case 'c':
                cpu_exec = true;
                break;
            case 't':
                type = atoi(optarg);
                break;
            case 'h':
            default:
                printHelp();
                return 0;
        }
    }


    if (!strcmp(action, (const char*)"test")) {
        run_test(x, y, type, cpu_exec);
        return 0;
    }

    if (!input0|| !expected) {
        printf("Invalid inputs.\n");
        return -1;
    }


    printf("input file: %s\n", input0);
    printf("solution fil: %s\n", expected);

    printf("Reading input file...\n");
    float *floatTemp = (float *)wbImport(input0, &numRowsA, &numColsA);
    hostA = (HAMC_DATA_TYPE_t *)malloc(numRowsA*numColsA * sizeof(HAMC_DATA_TYPE_t));
    for(int i = 0; i < numColsA * numRowsA; i++){
        hostA[i] = (HAMC_DATA_TYPE_t)floatTemp[i];
    }
    A = mat_init_cpu(numRowsA, numColsA);
    A->data = hostA;

    printf("Reading Solution file...\n");
    float *floatTemp2 = (float *)wbImport(expected, &numRowsS, &numColsS);
    sol = (HAMC_DATA_TYPE_t *)malloc(numRowsS*numColsS * sizeof(HAMC_DATA_TYPE_t));
    for(int i = 0; i < numColsS * numRowsS; i++){
        sol[i] = (HAMC_DATA_TYPE_t)floatTemp2[i];
    }


    printf("Input matrix:\n");
    printf("%d x %d\n", numRowsA, numColsA);
    for (int i = 0; i < numRowsA; i++) {
        for (int j = 0; j < numColsA; j++) {
            printf("%hu ",A->data[i*j + j]);
        }
        printf("\n");
    }

    if(cpu_exec) {
        printf("C Based execution:\n");
        B = transpose_cpu(A);
    }
    else {
        printf("GPU Based execution:\n");
        B = run_transpose_kernel(A);
    }


    printf("\n");
    printf("Solution matrix:\n");
    printf("%d x %d\n", numRowsS, numColsS);
    for (int i = 0; i < numRowsS; i++) {
        for (int j = 0; j < numColsS; j++) {
            printf("%hu ",sol[i*j + j]);
        }
        printf("\n");
    }
    printf("\n");


    printf("Output matrix:\n");
    printf("%d x %d\n", numRowsS, numColsS);
    for (int i = 0; i < numRowsS; i++) {
        for (int j = 0; j < numColsS; j++) {
            printf("%hu ",B->data[i*j + j]);
        }
        printf("\n");
    }


    if(B->rows != numRowsS && B->cols != numColsS){
        solved = false;
    }
    else{
        for(int i = 0; i < numRowsS * numColsS; i++){
            if(B->data[i] != sol[i]){
                std::cout << "i: " << i << std::endl;
                std::cout << "C->data[i]: " << B->data[i] << std::endl;
                std::cout << "expected: " << sol[i] << std::endl;
                solved = false;
                break;
            }
        }
    }

    std::cout << "solved: " << solved << std::endl;

    free(A);
    free(B);

    return 0;
}
