#include <hip/hip_runtime.h>
#include<stdlib.h>
#include <wb.h>
#include <stdint.h>
#include <ctype.h>
#include <stdio.h>
#include <unistd.h>


#include "../../hamc/hamc_common.h"
#include "../../hamc/hamc_cpu_code.c"

#include "MatrixAdd_cpu.c"
#include "../../hamc/MatrixAdd.cu"

#define TILE_WIDTH 16

#define CUDA_CHECK(ans)                                                   \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}

void printHelp()
{
    printf("run this executable with the following flags\n");
    printf("\n");
    printf("\t-i <input file name>\n");
    printf("\t-o <output file name>\n");
    printf("\t-s <solution file name>\n");
}


bin_matrix run_cpu(const char *in, HAMC_DATA_TYPE_t *sol)
{

    int numARows, numAColumns;


    HAMC_DATA_TYPE_t *hostA =
        (HAMC_DATA_TYPE_t *)wbImport(in, &numARows, &numAColumns);

    HAMC_DATA_TYPE_t *hostB =
        (HAMC_DATA_TYPE_t *)malloc(numARows*numAColumns * sizeof(HAMC_DATA_TYPE_t));

    bin_matrix A = mat_init_cpu(numARows, numAColumns);
    A->data = hostA;

    bin_matrix B = mat_init_cpu(numARows, numAColumns);
    B->data = hostB;


    //matrix_add(hostA, hostC, numARows, numAColumns);
    bin_matrix C = add_matrix_cpu(A, B);
    return C;
}


int main(int argc, char *argv[])
{
    printf("MatrixAdd test:\n");
    wbArg_t args;

    HAMC_DATA_TYPE_t *hostA; // The A matrix
    HAMC_DATA_TYPE_t *hostC; // The output C matrix
    HAMC_DATA_TYPE_t *deviceA; // A matrix on device
    HAMC_DATA_TYPE_t *deviceB; // B matrix on device (copy of A)
    HAMC_DATA_TYPE_t *deviceC; // C matrix on device
    int numARows;    // number of rows in the matrix A
    int numAColumns; // number of columns in the matrix A

    char *inputFileName;
    char *solutionFileName;

    hipEvent_t astartEvent, astopEvent;
    float aelapsedTime;
    hipEventCreate(&astartEvent);
    hipEventCreate(&astopEvent);

    int c;
    opterr = 0;
    printf("1\n");
    while ((c = getopt (argc, argv, "i:s:h")) != -1)
        switch(c)
        {
            case 'i':
                inputFileName = strdup(optarg);
                break;
            case 's':
                solutionFileName = strdup(optarg);
                break;
            case 'h':
                printHelp();
                return 0;
            default:
                abort();
        }

    args = wbArg_read(argc, argv);


    printf("input file: %s\n", inputFileName);
    printf("solution file: %s\n", solutionFileName);


    wbTime_start(Compute, "Performing CPU computation for MatrixAdd");
    run_cpu(inputFileName, solutionFileName);
    wbTime_stop(Compute, "Performing CPU computation");


    /* allocate host data for matrix */
    wbTime_start(Generic, "Importing data and creating memory on host");
    hostA = (HAMC_DATA_TYPE_t *)wbImport(inputFileName, &numARows, &numAColumns);
    int numBRows = numARows;    // number of rows in the matrix B
    int numBColumns = numAColumns; // number of columns in the matrix B
    int numCRows = numARows;    // number of rows in the matrix C
    int numCColumns = numAColumns; // number of columns in the matrix C
    hostC = (HAMC_DATA_TYPE_t *)malloc(numCRows*numCColumns * sizeof(HAMC_DATA_TYPE_t));
    wbTime_stop(Generic, "Importing data and creating memory on host");


    wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);


    /* allocate the memory space on GPU */
    wbTime_start(GPU, "Allocating GPU memory.");
    hipMalloc((void**) &deviceA, numARows * numAColumns * sizeof(HAMC_DATA_TYPE_t));
    hipMalloc((void**) &deviceB, numBRows * numBColumns * sizeof(HAMC_DATA_TYPE_t));
    hipMalloc((void**) &deviceC, numCRows * numCColumns * sizeof(HAMC_DATA_TYPE_t));
    wbTime_stop(GPU, "Allocating GPU memory.");


    dim3 dimGrid((numCColumns - 1) / 16 + 1, (numCRows - 1) / TILE_WIDTH + 1, 1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

    /* call CUDA kernel to perform computations */
    wbTime_start(Compute, "Performing CUDA computation for RREF");
    MatrixAdd_kernel<<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC, numARows);
    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");



    wbTime_start(Copy, "Copying output memory to the CPU");
    hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(HAMC_DATA_TYPE_t), hipMemcpyDeviceToHost);
    wbTime_stop(Copy, "Copying output memory to the CPU");

    /* Free GPU Memory */
    wbTime_start(GPU, "Freeing GPU Memory");
    hipFree(deviceA);
    hipFree(deviceC);
    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostC, numCRows, numCColumns);

    free(hostA);
    free(hostB);
    free(hostC);

    return 0;
}

