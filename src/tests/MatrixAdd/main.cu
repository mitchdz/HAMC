#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <wb.h>
#include <stdint.h>
#include <ctype.h>
#include <stdio.h>
#include <unistd.h>
#include <iostream>
#include <time.h>

#include "../../hamc/hamc_cpu_code.c"
#include "../../hamc/hamc_common.h"
#include "../../hamc/MatrixAdd.cu"

#define TILE_WIDTH 32 //32 //16

#define CUDA_CHECK(ans)                                                   \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}

#define mat_element(mat, row_idx, col_idx) \
    mat->data[row_idx * (mat->cols) + col_idx]

void printHelp()
{
    printf("run this executable with the following flags\n");
    printf("\n");
    printf("\t-i <input file name>\n");
    printf("\t-o <output file name>\n");
    printf("\t-s <solution file name>\n");
}

bin_matrix run_cpu(bin_matrix A, bin_matrix B)
{
    
    if (A->rows != B->rows || A->cols != B->cols){
        printf("Matrices are incompatible, check dimensions...\n");
        exit(0);
    }
    
    // Run CPU Operation
    bin_matrix hostC = add_matrix_cpu(A, B);

    
    return hostC;
}

static HAMC_DATA_TYPE_t *generate_data(int height, int width)
{
    HAMC_DATA_TYPE_t *data = (HAMC_DATA_TYPE_t *)malloc(sizeof(HAMC_DATA_TYPE_t) * width * height);
    int i;
    for (i = 0; i < width * height; i++) {
        data[i] = (HAMC_DATA_TYPE_t)(rand() % 2); // 0 or 1
    }
    return data;
}


bin_matrix run_kernel(bin_matrix A, bin_matrix B)
{
    if (A->rows != B->rows || A->cols != B->cols){
        printf("Matrices are incompatible, check dimensions...\n");
        exit(0);
    }

    //hipEvent_t astartEvent, astopEvent;
    //float aelapsedTime;
    //hipEventCreate(&astartEvent);
    //hipEventCreate(&astopEvent);
    
    
    
    HAMC_DATA_TYPE_t *deviceA;
    HAMC_DATA_TYPE_t *deviceB;
    HAMC_DATA_TYPE_t *deviceC;
    

    
    bin_matrix C = mat_init_cpu(A->rows,B->cols);

    //C->rows = A->rows;

    //C->cols = B->cols;

    
    /* allocate the memory space on GPU */
  //  wbTime_start(GPU, "Allocating GPU memory.");
    hipMalloc((void **) &deviceA, A->cols * A->rows * sizeof(HAMC_DATA_TYPE_t));
    hipMalloc((void **) &deviceB, B->cols * B->rows * sizeof(HAMC_DATA_TYPE_t));
    hipMalloc((void **) &deviceC, B->cols * A->rows * sizeof(HAMC_DATA_TYPE_t));
   // wbTime_stop(GPU, "Allocating GPU memory.");
    

    
    hipMemcpy(deviceA, A->data, A->cols * A->rows * sizeof(HAMC_DATA_TYPE_t), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, B->data, B->cols * B->rows * sizeof(HAMC_DATA_TYPE_t), hipMemcpyHostToDevice);
    
    printf("TILE_WIDTH -> %i \n", TILE_WIDTH);
    dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    int x_blocks = ((B->cols - 1)/TILE_WIDTH) + 1;
    int y_blocks = ((A->rows - 1)/TILE_WIDTH) + 1;
    dim3 DimGrid(x_blocks, y_blocks, 1);
   
   // hipEventRecord(astartEvent, 0);
    MatrixAdd<<<DimGrid, DimBlock>>>(deviceA, deviceB, deviceC, A->rows, A->cols);
    //hipDeviceSynchronize();
    //hipEventRecord(astopEvent, 0);
   // hipEventSynchronize(astopEvent);
   // hipEventElapsedTime(&aelapsedTime, astartEvent, astopEvent);
    
    //printf("\n");
    //printf("Total compute time (ms) %f for Matrix Add GPU\n\n",aelapsedTime);
    //printf("\n");
    
   // wbTime_start(Copy, "Copying output memory to the CPU");
    hipMemcpy(C->data, deviceC, B->cols * A->rows * sizeof(HAMC_DATA_TYPE_t), hipMemcpyDeviceToHost);
    //wbTime_stop(Copy, "Copying output memory to the CPU");
    
    //wbTime_start(GPU, "Freeing GPU Memory");
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
    //wbTime_stop(GPU, "Freeing GPU Memory");
   
    return C;
}

void run_test(int x, int y)
{

    clock_t start, end;
    double cpu_time_used;
    
    
    printf("X var = %i \n", x);
    printf("Y var = %i \n", y);
    
    // Matrix A
    HAMC_DATA_TYPE_t *raw_data0 = (HAMC_DATA_TYPE_t *)malloc(sizeof(HAMC_DATA_TYPE_t) * x * y);
    raw_data0 = generate_data(x, y);

    bin_matrix input0 = mat_init_cpu(x,y);
    input0->data = raw_data0;
    
    //Matrix B
    HAMC_DATA_TYPE_t *raw_data1 = (HAMC_DATA_TYPE_t *)malloc(sizeof(HAMC_DATA_TYPE_t) * x * y);
    raw_data1 = generate_data(x, y);

    bin_matrix input1 = mat_init_cpu(x,y);
    input1->data = raw_data1;
    
    

    /* CPU execution time */
        start = clock();

        bin_matrix CPU_BIN = run_cpu(input0, input1);

        end = clock();
        
        cpu_time_used = ((double) (end - start))/ CLOCKS_PER_SEC;
        printf("CPU time: %lf \n", cpu_time_used);
    

    /* GPU execution time */
    start = clock();

    bin_matrix GPU_BIN = run_kernel(input0, input1);

    end = clock();
    cpu_time_used = ((double) (end - start))/ CLOCKS_PER_SEC;
    printf("GPU time: %lf \n", cpu_time_used);
    
    char * tester = "SUCCESS!";
    
    for(int i = 0; i < x*y; i++)
    {
    	if(GPU_BIN->data[i] != CPU_BIN->data[i])
    	{
    	   tester = "FAIL!";
    	   printf("Index: %i \n", i);
    	   printf("Kernel Output: %i \n", GPU_BIN->data[i]);
    	   printf("Expected: %i \n", CPU_BIN->data[i]);
    	}
    }
    
    printf("The test was a: %s!\n", tester);

    free(input0);
    free(input1);
    free(raw_data0);
    free(raw_data1);
    free(CPU_BIN);
    free(GPU_BIN);

}

// main - should only be handling the initial matrices A and B generation and input files
int main(int argc, char *argv[])
{
    

    // Variables - Rows & Cols
    int numARows;    // number of rows in the matrix A
    int numAColumns; // number of columns in the matrix A
    int numBRows;    // number of rows in the matrix B
    int numBColumns; // number of columns in the matrix B
    
 
    // Inputs
    wbArg_t args;
    
    // Output
    bin_matrix hostC;
    
 
    char *inputFileName0;
    char *inputFileName1;
    char *solutionFileName;
    

    bool cpu_run = false;
    char *just_test = NULL;
    
    int x, y;

    hipEvent_t astartEvent, astopEvent;
    float aelapsedTime;
    
    // Give Input Files
   
    int c;
    opterr = 0;
    while ((c = getopt (argc, argv, "t:x:y:i:j:s:lhp")) != -1){
        switch(c)
        {
            case 't':
            	just_test = strdup(optarg);
            	break;
            case 'x':
            	
            	x = atoi(optarg);
            	break;
            case 'y':
            	
            	y = atoi(optarg);
            	break;
            case 'i':
            	
                inputFileName0 = strdup(optarg);
                printf("i1 -> %s",inputFileName0);
                break;
            case 'j':
            	
                inputFileName1 = strdup(optarg);
                printf("i2 -> %s",inputFileName1);
                break;
            case 's':
            	
                solutionFileName = strdup(optarg);
                printf("o -> %s",solutionFileName);
                break;
            case 'h':
                printHelp();
                return 0;
            case 'n':
            case 'l':
            case 'p':
            	printf("p check \n\n");
                cpu_run = true;
                break;
            default:
                abort();
        }
   }
    
    
    if (!strcmp(just_test, (const char*)"test")) {
    	printf("Test is running! \n");
        run_test(x, y);
        return 0;
    } //else {
    	//printf("Input Files are running! \n");
    	//args = wbArg_read(argc, argv);
    //}
    
    // Read Input Files
        // Input File 0 - Matrix A
    
    float *hostAFloats = (float *)wbImport(inputFileName0, &numARows, &numAColumns);
    HAMC_DATA_TYPE_t *hostA = (HAMC_DATA_TYPE_t *)malloc(numARows*numAColumns * sizeof(HAMC_DATA_TYPE_t));
    for (int i = 0; i < numARows*numAColumns; i++)
    {
        hostA[i] = (HAMC_DATA_TYPE_t)hostAFloats[i];
    }
    
        // Input File 1 - Matrix B
    float *hostBFloats = (float *)wbImport(inputFileName1, &numBRows, &numBColumns);
    
    HAMC_DATA_TYPE_t *hostB = (HAMC_DATA_TYPE_t *)malloc(numBRows*numBColumns * sizeof(HAMC_DATA_TYPE_t));
   
    for (int i = 0; i < numBRows*numBColumns; i++)
    {
        hostB[i] = (HAMC_DATA_TYPE_t)hostBFloats[i];
    }
    
    // Input File 1 - Solution File
    float *hostOutputFile = (float *)wbImport(solutionFileName, &numBRows, &numBColumns);
    
    HAMC_DATA_TYPE_t *hostOutput = (HAMC_DATA_TYPE_t *)malloc(numBRows*numBColumns * sizeof(HAMC_DATA_TYPE_t));
   
    for (int i = 0; i < numBRows*numBColumns; i++)
    {
        hostOutput[i] = (HAMC_DATA_TYPE_t)hostOutputFile[i];
    }
    
    
    printf("8 \n\n\n");  
    // Initilizing the Matricies HERE!
    
    // Matrix A
    
    bin_matrix hostABin = mat_init_cpu(numARows, numAColumns);
    
    for (int i = 0; i < numARows*numAColumns; i++) 
    {
        hostABin->data[i] = hostA[i];
    }
    
    // Matrix B
  
    bin_matrix hostBBin = mat_init_cpu(numBRows, numBColumns);
    
    for (int i = 0; i < numBRows*numBColumns; i++) 
    {
       hostBBin->data[i] = hostB[i];
    }  
    
    // Call Either Kernels
    if(cpu_run)
    {
        
        hostC = run_cpu(hostABin, hostBBin);
        
    } else {
        hostC = run_kernel(hostABin,hostBBin);
        
    }
        
    // Check Solution
    
   
    //wbSolution(args, hostC, numARows, numAColumns);
    
    for(int i = 0; i < numARows*numAColumns; i++)
    {
//    	if(hostC->data[i] != hostOutput[i])
//    	{
//    	   printf("Index: %i \n", i);
//    	   printf("Kernel Output: %i \n", hostC->data[i]);
//    	   printf("Expected: %i \n", hostOutput[i]);
//    	}
//    	printf("%i \n", i);
//        printf("hostC->data[%i] -> %i \n",i, hostC->data[i]);
//        printf("hostOutput[%i] -> %i \n",i, hostOutput[i]);
    }

    
    free(hostABin);
    free(hostBBin);
    free(hostA);
    free(hostB);
    free(hostC);

    return 0;
}
