#include <hip/hip_runtime.h>
#include<stdlib.h>
#include <wb.h>
#include <stdint.h>
#include <ctype.h>
#include <stdio.h>
#include <unistd.h>

#include "MatrixAdd_cpu.h"
#include "../../hamc/MatrixAdd.cu"

#define TILE_WIDTH 16

#define CUDA_CHECK(ans)                                                   \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}

void printHelp()
{
    printf("run this executable with the following flags\n");
    printf("\n");
    printf("\t-i <input file name>\n");
    printf("\t-o <output file name>\n");
    printf("\t-s <solution file name>\n");
}


void run_cpu(const char *in, const char*sol)
{
    int numARows, numAColumns;
    ushort *hostA = (ushort *)wbImport(in, &numARows, &numAColumns);
    ushort *hostC = (ushort *)malloc(numARows*numAColumns * sizeof(ushort));

    matrix_add(hostA, hostC, numARows, numAColumns);
}


int main(int argc, char *argv[])
{
    printf("MatrixAdd test:\n");
    wbArg_t args;

    ushort *hostA; // The A matrix
    ushort *hostC; // The output C matrix
    ushort *deviceA; // A matrix on device
    ushort *deviceB; // B matrix on device (copy of A)
    ushort *deviceC; // C matrix on device
    int numARows;    // number of rows in the matrix A
    int numAColumns; // number of columns in the matrix A

    char *inputFileName;
    char *solutionFileName;

    hipEvent_t astartEvent, astopEvent;
    float aelapsedTime;
    hipEventCreate(&astartEvent);
    hipEventCreate(&astopEvent);

    int c;
    opterr = 0;
    printf("1\n");
    while ((c = getopt (argc, argv, "i:s:h")) != -1)
        switch(c)
        {
            case 'i':
                inputFileName = strdup(optarg);
                break;
            case 's':
                solutionFileName = strdup(optarg);
                break;
            case 'h':
                printHelp();
                return 0;
            default:
                abort();
        }

    args = wbArg_read(argc, argv);


    printf("input file: %s\n", inputFileName);
    printf("solution file: %s\n", solutionFileName);


    wbTime_start(Compute, "Performing CPU computation for MatrixAdd");
    run_cpu(inputFileName, solutionFileName);
    wbTime_stop(Compute, "Performing CPU computation");


    /* allocate host data for matrix */
    wbTime_start(Generic, "Importing data and creating memory on host");
    hostA = (ushort *)wbImport(inputFileName, &numARows, &numAColumns);
    int numBRows = numARows;    // number of rows in the matrix B
    int numBColumns = numAColumns; // number of columns in the matrix B
    int numCRows = numARows;    // number of rows in the matrix C
    int numCColumns = numAColumns; // number of columns in the matrix C
    hostC = (ushort *)malloc(numCRows*numCColumns * sizeof(ushort));
    wbTime_stop(Generic, "Importing data and creating memory on host");


    wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);


    /* allocate the memory space on GPU */
    wbTime_start(GPU, "Allocating GPU memory.");
    hipMalloc((void**) &deviceA, numARows * numAColumns * sizeof(ushort));
    hipMalloc((void**) &deviceB, numBRows * numBColumns * sizeof(ushort));
    hipMalloc((void**) &deviceC, numCRows * numCColumns * sizeof(ushort));
    wbTime_stop(GPU, "Allocating GPU memory.");


    dim3 dimGrid((numCColumns - 1) / 16 + 1, (numCRows - 1) / TILE_WIDTH + 1, 1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

    /* call CUDA kernel to perform computations */
    wbTime_start(Compute, "Performing CUDA computation for RREF");
    MatrixAdd_kernel<<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC, numARows);
    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");



    wbTime_start(Copy, "Copying output memory to the CPU");
    hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(ushort), hipMemcpyDeviceToHost);
    wbTime_stop(Copy, "Copying output memory to the CPU");

    /* Free GPU Memory */
    wbTime_start(GPU, "Freeing GPU Memory");
    hipFree(deviceA);
    hipFree(deviceC);
    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostC, numCRows, numCColumns);

    free(hostA);
    free(hostB);
    free(hostC);

    return 0;
}

