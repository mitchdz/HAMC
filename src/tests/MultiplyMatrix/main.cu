#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <wb.h>
#include <stdint.h>
#include <ctype.h>
#include <stdio.h>
#include <unistd.h>
#include <iostream>

#include "../../hamc/hamc_cpu_code.c"
#include "../../hamc/MultiplyMatrix.cu"

#define TILE_WIDTH 16
#define ushort unsigned short

#define CUDA_CHECK(ans)                                                   \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess){
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

void printHelp()
{
    printf("run this executable with the following flags\n");
    printf("\n");
    printf("\t-a <input0 file name>\n");
    printf("\t-b <input1 file name>\n");
    printf("\t-e <expected solution file name>\n");
    printf("\t-o <output file name>\n");
    printf("\t-c \n");
}


bin_matrix run_cpu(bin_matrix A, bin_matrix B)
{
    return matrix_mult_cpu(A, B);
}

bin_matrix run_kernel(bin_matrix A, bin_matrix B)
{
    if (A->cols != B->rows){
        printf("Matrices are incompatible, check dimensions...\n");
        exit(0);
    }

    ushort *deviceA;
    ushort *deviceB;
    ushort *deviceC;
    bin_matrix C = mat_init_cpu(A->rows, A->cols);
    
    hipMalloc((void **) &deviceA, A->cols * A->rows * sizeof(ushort));
    hipMalloc((void **) &deviceB, B->cols * B->rows * sizeof(ushort));
    hipMalloc((void **) &deviceC, B->cols * A->rows * sizeof(ushort));
    
    hipMemcpy(deviceA, A->data, A->cols * A->rows * sizeof(ushort), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, B->data, B->cols * B->rows * sizeof(ushort), hipMemcpyHostToDevice);
    
    dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    int x_blocks = ((B->cols - 1)/TILE_WIDTH) + 1;
    int y_blocks = ((A->rows - 1)/TILE_WIDTH) + 1;
    dim3 DimGrid(x_blocks, y_blocks, 1);
    //dim3 DimBlock(TILE_WIDTH * TILE_WIDTH, 1, 1);
    //dim3 DimGrid(((A->rows * B->cols) - 1) / (TILE_WIDTH * TILE_WIDTH), 1, 1);
    
    mult_kernel<<<DimGrid, DimBlock>>>(deviceA, deviceB, deviceC, A->rows, B->rows, A->cols, B->cols);
    
    hipDeviceSynchronize();
    
    hipMemcpy(C->data, deviceC, B->cols * A->rows * sizeof(ushort), hipMemcpyDeviceToHost);
    
    std::cout << "C->data";
    for(int i = 0; i < (C-rows * C->cols); i++){
        if(i % TILE_WIDTH == 0) std::cout << endl;
        std::cout << C->data[i];
    }
    std::cout << endl;
    
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    return C;
}

int main(int argc, char *argv[])
{
    wbArg_t args;
    bin_matrix A;
    bin_matrix B;
    bin_matrix C;
    int numRowsA;
    int numColsA;
    int numRowsB;
    int numColsB;
    int numRowsS;
    int numColsS;
    ushort *hostA;
    ushort *hostB;
    ushort *sol;
    char *input0;
    char *input1;
    char *expected;
    bool cpu_exec = false;
    bool solved = true;
    
    int opt;
    while ((opt = getopt(argc, argv, "a:b:e:o:c")) != -1){
        switch(opt){
            case 'a':
                input0 = strdup(optarg);
                break;
            case 'b':
                input1 = strdup(optarg);
                break;
            case 'e':
                expected = strdup(optarg);
                break;
            case 'o':
                //input0 = strdup(optarg);
                break;
            case 'c':
                cpu_exec = true;
                break;
            case 'h':
            default:
                printHelp();
                return 0;
        }
    }
    float *floatTemp = (float *)wbImport(input0, &numRowsA, &numColsA);
    hostA = (ushort *)malloc(numRowsA*numColsA * sizeof(ushort));
    for(int i = 0; i < numColsA * numRowsA; i++){
        hostA[i] = (ushort)floatTemp[i];
    }
    A = mat_init_cpu(numRowsA, numColsA);
    A->data = hostA;
    
    floatTemp = (float *)wbImport(input1, &numRowsB, &numColsB);
    hostB = (ushort *)malloc(numRowsB*numColsB * sizeof(ushort));
    for(int i = 0; i < numColsB * numRowsB; i++){
        hostB[i] = (ushort)floatTemp[i];
    }    
    B = mat_init_cpu(numRowsB, numColsB);
    B->data = hostB;
    
    floatTemp = (float *)wbImport(expected, &numRowsS, &numColsS);
    sol = (ushort *)malloc(numRowsS*numColsS * sizeof(ushort));
    for(int i = 0; i < numColsB * numRowsB; i++){
        sol[i] = (ushort)floatTemp[i];
    }    
    
    std::cout << "A->data";
    for(int i = 0; i < numColsA * numRowsA; i++){
        if(i%16 == 0) std::cout << "" << std::endl;
        std::cout << hostA[i] << " ";
    }
    std::cout << std::endl;
    std::cout << "B->data";
    for(int i = 0; i < numColsB * numRowsB; i++){
        if(i%16 == 0) std::cout << "" << std::endl;
        std::cout << hostB[i] << " ";
    }
    std::cout << std::endl;
    
    if(cpu_exec){
        C = run_cpu(A, B);
    }
    else{
        std::cout << "Running Kernel" << std::endl;
        C = run_kernel(A, B);
    }
    //C = (cpu_exec) ? run_cpu(A, B) : run_kernel(A, B);
    
    std::cout << "C->data";
    for(int i = 0; i < C->cols * C->rows; i++){
        if(i%16 == 0) std::cout << "" << std::endl;
        std::cout << C->data[i] << " ";
    }
    std::cout << std::endl;
    
    if(C->rows != numRowsS && C->cols != numColsS){
        solved = false;
    }
    else{
        for(int i = 0; i < numRowsS * numColsS; i++){
            if(C->data[i] != sol[i]){
                std::cout << "i: " << i << std::endl;
                std::cout << "C->data[i]: " << C->data[i] << std::endl;
                std::cout << "expected: " << sol[i] << std::endl;
                solved = false;
                break;
            }
        }
    }
    
    /*std::cout << "C->rows: " << C->rows << std::endl;
    std::cout << "C->cols: " << C->cols << std::endl;
    std::cout << "numRowsS: " << numRowsS << std::endl;
    std::cout << "numColsS: " << numColsS << std::endl;*/
    std::cout << "solved: " << solved << std::endl;
    
    free(A);
    free(B);
    free(C);
    
    return 0;
}