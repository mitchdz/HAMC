#include "hip/hip_runtime.h"
#ifndef HAMC_SCRATCH_H
#define HAMC_SCRATCH_H


#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <stdint.h>

#include <sys/time.h>


#include "../../src/hamc/hamc_cpu_code.c"

using namespace std;

#define BLOCK_SIZE_LU 16
#define BLOCK_SIZE_LU2 256

__global__ void ForwardSolve(HAMC_DATA_TYPE_t* A, HAMC_DATA_TYPE_t* b, int n, int k, int half_k, int i){
    int ty = threadIdx.y;
    int by = blockIdx.y;
    int tidy = by*BLOCK_SIZE_LU2+ty;
    int row = tidy + i + 1;
    __shared__ HAMC_DATA_TYPE_t mult;

    if(ty==0){
        mult = b[i];
    }

    __syncthreads();

    if(tidy < half_k && row < n){
        b[row] ^= A[row*k + half_k - 1 - tidy] & mult;
    }
}



__global__ void BackSolve(HAMC_DATA_TYPE_t* A, HAMC_DATA_TYPE_t* b, int n, int k, int half_k, int i){
  int ty = threadIdx.y;
  int by = blockIdx.y;
  int tidy = by*BLOCK_SIZE_LU2+ty;
  int row = i - 1 - tidy;
  __shared__ HAMC_DATA_TYPE_t mult;

  if(ty==0){
    b[i] = b[i]/A[i*k + half_k];
    mult = b[i];
  }

  __syncthreads();

  if(tidy < half_k && row >= 0){
    b[row] ^=  A[row*k + half_k + 1 + tidy] & mult;
  }

}


__global__ void add(HAMC_DATA_TYPE_t *A, HAMC_DATA_TYPE_t *B, HAMC_DATA_TYPE_t *C)
{
    int tid = blockIdx.x;

    C[tid] = A[tid] ^ B[tid];
}

__global__ void reduce(HAMC_DATA_TYPE_t *A, int size, int index, int b_size)
{
    extern __shared__ HAMC_DATA_TYPE_t pivot[];

    int i;

    int tid=threadIdx.x;
    int bid=blockIdx.x;
    int block_size=b_size;

    //int pivot_start=(index*size+index);
    //int pivot_end=(index*size+size);

    int start, end, pivot_row, my_row;

    if(tid==0){
        for(i=index;i<size;i++)  {
            pivot[i]=A[(index*size)+i];
        }
    }

    __syncthreads();

    pivot_row=(index * size);
    my_row=(((block_size * bid) + tid) * size);
    start=my_row + index;
    end=my_row + size;

    if(my_row >pivot_row){
        for(i=start+1 ; i < end;i++){
            //A[i]=A[i]-(A[start]*pivot[(i-my_row)]);
            A[i] ^= (A[start] & pivot[(i-my_row)]);
        }
    }
}

// 1) A = P * L * U
// 2) y*U = I // y is unkown
// 3) z*L = y // z is unkown
// 4) x*P = z // x is unkown, x is the inverse of A
int main(int argc, char *argv[]){
    printf("Scratch test\n");
    HAMC_DATA_TYPE_t *a;
    HAMC_DATA_TYPE_t *c;
    int N;
    int flag=0;

    HAMC_DATA_TYPE_t **result;
    HAMC_DATA_TYPE_t **b;
    int blocks;

    HAMC_DATA_TYPE_t *dev_a;
    int i;

    double start;
    double end;
    struct timeval tv;

    N = 3;

    //allocate memory on CPU
    a = (HAMC_DATA_TYPE_t *)malloc(sizeof(HAMC_DATA_TYPE_t)*N*N);
    c = (HAMC_DATA_TYPE_t *)malloc(sizeof(HAMC_DATA_TYPE_t)*N*N);


    result = (HAMC_DATA_TYPE_t **)malloc(sizeof(HAMC_DATA_TYPE_t *)*N);
    b = (HAMC_DATA_TYPE_t **)malloc(sizeof(HAMC_DATA_TYPE_t *)*N);


    for(i = 0; i < N; i++){
       result[i]=(HAMC_DATA_TYPE_t *)malloc(sizeof(HAMC_DATA_TYPE_t)*N);
       b[i]     =(HAMC_DATA_TYPE_t *)malloc(sizeof(HAMC_DATA_TYPE_t)*N);
    }

    //allocate the memory on the GPU
    hipMalloc ( (void**)&dev_a, N*N*sizeof (HAMC_DATA_TYPE_t) );

    bin_matrix sol_raw = mat_init_cpu(N,N);

    srand((unsigned)2);
    //fill the arrays 'a' on the CPU
    for ( i = 0; i <= (N*N); i++) {
        HAMC_DATA_TYPE_t val = ((rand()%2));
        a[i] = val;
        sol_raw->data[i] = val;
    }

    printf("Matrix a is :\n");
    for(i=0; i<(N*N); i++){
        if(i%N==0)
            printf("\n %d ", a[i]);
        else
            printf("%d ",a[i]);
    }
    printf("\n\n");

    bin_matrix sol = circ_matrix_inverse_cpu(sol_raw);

    printf("Expected solution is :\n");
    for(i=0; i<(N*N); i++){
        if(i%N==0)
            printf("\n %d ", sol->data[i]);
        else
            printf("%d ",sol->data[i]);
    }

    hipMemcpy(dev_a,a,N*N*sizeof(HAMC_DATA_TYPE_t), hipMemcpyHostToDevice);//copy array to device memory

    gettimeofday(&tv,NULL);
    start=tv.tv_sec;


    /* LU decomposition */
    for(i = 0; i < N; i++){
        blocks=((N/512));
        reduce<<<blocks,512,N*sizeof(HAMC_DATA_TYPE_t)>>>(dev_a,N,i,512);
    }

    // 1) A = P * L * U
    // 2) y*U = I // y is unkown
    // 3) z*L = y // z is unkown
    // 4) x*P = z // x is unkown, x is the inverse of A

    gettimeofday(&tv,NULL);
    end=tv.tv_sec;
    hipMemcpy( c, dev_a, N*N*sizeof(HAMC_DATA_TYPE_t),hipMemcpyDeviceToHost );//copy array back to host

    printf("\nThe time for LU decomposition is %lf \n",(end-start));
       //display the results


    printf("Output from GPU is \n");
    for ( i = 0; i < (N*N); i++) {
               if(i%N==0)
        printf( "\n%d  ", c[i]);
               else  printf("%d ",c[i]);
    }
    printf("\n");



    printf("Performing Forward and backwards substition\n");


    for ( i = 0; i < N; i++) {

        for (int j = 0; j < N; j++) {
            // Forward solve
        }

        for (int j = N - 1; j >= 0; j++) {
            // Backwards solve


        }

    }


    hipMemcpy(c, dev_a, N*N*sizeof(HAMC_DATA_TYPE_t), hipMemcpyDeviceToHost);



    printf("final result:\n");
    for ( i = 0; i < N; i++) {
        for ( int j = 0; j < N; j++) {
            printf("%d ", c[i*N + j]);
        }
        printf("\n");
    }

    // check results
    for (int i = 0; i < N*N; i++) {
        if (sol->data[i] != c[i]) {
            flag = 1;
            break;
        }
    }

    if(flag==0) printf("correctq: Correct");
    else printf("correctq: Failure %d \n",flag);

    // free the memory allocated on the GPU
    hipFree( dev_a );

    return 0;
}


#endif /* HAMC_SCRATCH_H */